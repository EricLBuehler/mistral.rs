#include "hip/hip_runtime.h"
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <stdint.h>

#include "cuda_compat.h"

namespace vllm {

template <typename scalar_t, bool IS_NEOX>
inline __device__ void apply_rotary_embedding(
    scalar_t *__restrict__ arr, const scalar_t *__restrict__ cos_ptr,
    const scalar_t *__restrict__ sin_ptr, int rot_offset, int rot_dim) {
  int x_index, y_index;
  scalar_t cos, sin;
  if (IS_NEOX) {
    // GPT-NeoX style rotary embedding.
    x_index = rot_offset;
    y_index = rot_dim + rot_offset;
    cos = VLLM_LDG(cos_ptr + x_index);
    sin = VLLM_LDG(sin_ptr + x_index);
  } else {
    // GPT-J style rotary embedding.
    x_index = 2 * rot_offset;
    y_index = 2 * rot_offset + 1;
    cos = VLLM_LDG(cos_ptr + x_index / 2);
    sin = VLLM_LDG(sin_ptr + x_index / 2);
  }

  const scalar_t x = arr[x_index];
  const scalar_t y = arr[y_index];
  arr[x_index] = x * cos - y * sin;
  arr[y_index] = y * cos + x * sin;
}

template <typename scalar_t, bool IS_NEOX>
__global__ void rotary_embedding_kernel(
    scalar_t *__restrict__ query, // [num_tokens, num_heads, head_size]
    scalar_t *__restrict__ key,   // [num_tokens, num_heads, head_size]
    const scalar_t *__restrict__ cos_cache, // [num_tokens, rot_dim]
    const scalar_t *__restrict__ sin_cache, // [num_tokens, rot_dim]
    const int rot_dim, const int64_t query_stride, const int64_t key_stride,
    const int num_heads, const int num_kv_heads, const int head_size) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;

  const scalar_t *cos_ptr = cos_cache + token_idx * rot_dim;
  const scalar_t *sin_ptr = sin_cache + token_idx * rot_dim;

  const int nq = num_heads * rot_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / rot_dim;
    const int64_t token_head = token_idx * query_stride + head_idx * head_size;
    const int rot_offset = i % rot_dim;
    apply_rotary_embedding<scalar_t, IS_NEOX>(query + token_head, cos_ptr,
                                              sin_ptr, rot_offset, rot_dim);
  }

  const int nk = num_kv_heads * rot_dim;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / rot_dim;
    const int64_t token_head = token_idx * key_stride + head_idx * head_size;
    const int rot_offset = i % rot_dim;
    apply_rotary_embedding<scalar_t, IS_NEOX>(key + token_head, cos_ptr,
                                              sin_ptr, rot_offset, rot_dim);
  }
}

} // namespace vllm

#define CALL_ROTARY(T, IS_NEOX)                                                \
  vllm::rotary_embedding_kernel<T, IS_NEOX><<<grid, block, 0, stream>>>(       \
      reinterpret_cast<T *>(query), reinterpret_cast<T *>(key),                \
      reinterpret_cast<T *>(cos_cache), reinterpret_cast<T *>(sin_cache),      \
      rot_dim, query_stride, key_stride, num_heads, num_kv_heads, head_size);

extern "C" void
rotary_embedding(void *query,     // [num_tokens, num_heads, head_size]
                 void *key,       // [num_tokens, num_kv_heads, head_size]
                 void *cos_cache, // [num_tokens, rot_dim]
                 void *sin_cache, // [num_tokens, rot_dim]
                 int32_t is_neox,

                 int32_t head_size, int64_t num_tokens, int32_t rot_dim,
                 int32_t num_heads, int32_t num_kv_heads, int64_t query_stride,
                 int64_t key_stride,

                 uint32_t dtype // 0 => f16; 1 => bf16; 2 => f32
) {

  dim3 grid(num_tokens);
  dim3 block(std::min(num_heads * rot_dim, 512));
  const hipStream_t stream = 0;
  const bool is_neox_bool = is_neox;

  if (is_neox_bool) {
    if (dtype == 0) {
      CALL_ROTARY(half, true);
    } else if (dtype == 1) {
      CALL_ROTARY(__hip_bfloat16, true);
    } else if (dtype == 2) {
      CALL_ROTARY(float, true);
    }
  } else {
    if (dtype == 0) {
      CALL_ROTARY(half, false);
    } else if (dtype == 1) {
      CALL_ROTARY(__hip_bfloat16, false);
    } else if (dtype == 2) {
      CALL_ROTARY(float, false);
    }
  }
}