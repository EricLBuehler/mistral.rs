#include <cstdint>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>

#define CUDA_CHECK(call)                                                       \
  do {                                                                         \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__,         \
              hipGetErrorString(err));                                        \
      exit(err);                                                               \
    }                                                                          \
  } while (0)

template <typename T>
__global__ void dequant_fp8_blockwise_kernel(
    const __hip_fp8_e4m3_fnuz *__restrict__ weight, const float *__restrict__ scale,
    T *__restrict__ output, int weight_height, int weight_width,
    int weight_row_stride, int scale_stride, int weight_block_size_y,
    int weight_block_size_x) {
  // Each block corresponds to a tile.
  int grid_y = blockIdx.y; // tile row index
  int grid_x = blockIdx.x; // tile column index

  // Compute the starting indices for this tile.
  int start_y = grid_y * weight_block_size_y;
  int start_x = grid_x * weight_block_size_x;

  // Load the block's scale factor into shared memory.
  __shared__ float block_scale;
  if (threadIdx.x == 0 && threadIdx.y == 0) {
    block_scale = scale[grid_y * scale_stride + grid_x];
  }
  __syncthreads(); // Ensure all threads see the loaded value.

  // Loop over the tile using a fixed blockDim, covering the whole tile.
  for (int local_y = threadIdx.y; local_y < weight_block_size_y;
       local_y += blockDim.y) {
    for (int local_x = threadIdx.x; local_x < weight_block_size_x;
         local_x += blockDim.x) {
      int weight_y = start_y + local_y;
      int weight_x = start_x + local_x;
      if (weight_y < weight_height && weight_x < weight_width) {
        int pos = weight_y * weight_row_stride + weight_x;
        float w_val =
            __half2float(__hip_cvt_fp8_to_halfraw(weight[pos].__x, __HIP_E4M3_FNUZ));
        output[pos] = static_cast<T>(w_val * block_scale);
      }
    }
  }
}

extern "C" void launch_dequant_fp8_blockwise_kernel_f32(
    const __hip_fp8_e4m3_fnuz *d_weight, const float *d_scale, float *d_output,
    int weight_height, int weight_width, int weight_row_stride,
    int scale_stride, int weight_block_size_y, int weight_block_size_x,
    hipStream_t stream) {
  int grid_y = (weight_height + weight_block_size_y - 1) / weight_block_size_y;
  int grid_x = (weight_width + weight_block_size_x - 1) / weight_block_size_x;
  dim3 blockDim(32, 32);
  dim3 gridDim(grid_x, grid_y);

  dequant_fp8_blockwise_kernel<float><<<gridDim, blockDim, 0, stream>>>(
      d_weight, d_scale, d_output, weight_height, weight_width,
      weight_row_stride, scale_stride, weight_block_size_y,
      weight_block_size_x);
  CUDA_CHECK(hipGetLastError());
}

extern "C" void launch_dequant_fp8_blockwise_kernel_f16(
    const __hip_fp8_e4m3_fnuz *d_weight, const float *d_scale, __half *d_output,
    int weight_height, int weight_width, int weight_row_stride,
    int scale_stride, int weight_block_size_y, int weight_block_size_x,
    hipStream_t stream) {
  int grid_y = (weight_height + weight_block_size_y - 1) / weight_block_size_y;
  int grid_x = (weight_width + weight_block_size_x - 1) / weight_block_size_x;
  dim3 blockDim(32, 32);
  dim3 gridDim(grid_x, grid_y);

  dequant_fp8_blockwise_kernel<__half><<<gridDim, blockDim, 0, stream>>>(
      d_weight, d_scale, d_output, weight_height, weight_width,
      weight_row_stride, scale_stride, weight_block_size_y,
      weight_block_size_x);
  CUDA_CHECK(hipGetLastError());
}

extern "C" void launch_dequant_fp8_blockwise_kernel_bf16(
    const __hip_fp8_e4m3_fnuz *d_weight, const float *d_scale,
    __hip_bfloat16 *d_output, int weight_height, int weight_width,
    int weight_row_stride, int scale_stride, int weight_block_size_y,
    int weight_block_size_x, hipStream_t stream) {
  int grid_y = (weight_height + weight_block_size_y - 1) / weight_block_size_y;
  int grid_x = (weight_width + weight_block_size_x - 1) / weight_block_size_x;
  dim3 blockDim(32, 32);
  dim3 gridDim(grid_x, grid_y);

  dequant_fp8_blockwise_kernel<__hip_bfloat16><<<gridDim, blockDim, 0, stream>>>(
      d_weight, d_scale, d_output, weight_height, weight_width,
      weight_row_stride, scale_stride, weight_block_size_y,
      weight_block_size_x);
  CUDA_CHECK(hipGetLastError());
}