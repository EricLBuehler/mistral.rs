#include <cassert>
#include <cstdint>
#include <hip/hip_runtime.h>

extern "C" void launch_dequant_fp8_blockwise_kernel_f32(
    const uint8_t *d_weight, const float *d_scale, float *d_output,
    int weight_height, int weight_width, int weight_row_stride,
    int scale_stride, int weight_block_size_y, int weight_block_size_x,
    hipStream_t stream) {
  assert(false);
}

extern "C" void launch_dequant_fp8_blockwise_kernel_f16(
    const uint8_t *d_weight, const float *d_scale, uint16_t *d_output,
    int weight_height, int weight_width, int weight_row_stride,
    int scale_stride, int weight_block_size_y, int weight_block_size_x,
    hipStream_t stream) {
  assert(false);
}

extern "C" void launch_dequant_fp8_blockwise_kernel_bf16(
    const uint8_t *d_weight, const float *d_scale, uint16_t *d_output,
    int weight_height, int weight_width, int weight_row_stride,
    int scale_stride, int weight_block_size_y, int weight_block_size_x,
    hipStream_t stream) {
  assert(false);
}

extern "C" void launch_quant_fp8_blockwise_kernel_f32(
    const float *d_input, uint8_t *d_weight, float *d_scale, int weight_height,
    int weight_width, int weight_row_stride, int scale_stride,
    int weight_block_size_y, int weight_block_size_x, hipStream_t stream) {
  assert(false);
}

extern "C" void launch_quant_fp8_blockwise_kernel_f16(
    const uint16_t *d_input, uint8_t *d_weight, float *d_scale,
    int weight_height, int weight_width, int weight_row_stride,
    int scale_stride, int weight_block_size_y, int weight_block_size_x,
    hipStream_t stream) {
  assert(false);
}

extern "C" void launch_quant_fp8_blockwise_kernel_bf16(
    const uint16_t *d_input, uint8_t *d_weight, float *d_scale,
    int weight_height, int weight_width, int weight_row_stride,
    int scale_stride, int weight_block_size_y, int weight_block_size_x,
    hipStream_t stream) {
  assert(false);
}