#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <assert.h>
#include <iostream>
#include "marlin/marlin.cuh"
#include "marlin/marlin_dtypes.cuh"
using namespace marlin;

template <int const num_threads, int const num_bits, bool const has_perm>
__global__ void
gptq_marlin_repack_kernel(uint32_t const *__restrict__ b_q_weight_ptr,
                          uint32_t const *__restrict__ perm_ptr,
                          uint32_t *__restrict__ out_ptr, int size_k,
                          int size_n) {
  constexpr int pack_factor = 32 / num_bits;

  int k_tiles = size_k / tile_k_size;
  int n_tiles = size_n / tile_n_size;
  int block_k_tiles = div_ceil(k_tiles, gridDim.x);

  int start_k_tile = blockIdx.x * block_k_tiles;
  if (start_k_tile >= k_tiles) {
    return;
  }

  int finish_k_tile = min(start_k_tile + block_k_tiles, k_tiles);

  // Wait until the next thread tile has been loaded to shared memory.
  auto wait_for_stage = [&]() {
    // We only have `stages - 2` active fetches since we are double buffering
    // and can only issue the next fetch when it is guaranteed that the previous
    // shared memory load is fully complete (as it may otherwise be
    // overwritten).
    cp_async_wait<repack_stages - 2>();
    __syncthreads();
  };

  extern __shared__ int4 sh[];

  constexpr int perm_size = tile_k_size / 4;

  int4 *sh_perm_ptr = sh;
  int4 *sh_pipe_ptr = sh_perm_ptr;
  if constexpr (has_perm) {
    sh_pipe_ptr += perm_size;
  }

  constexpr int tile_ints = tile_k_size / pack_factor;

  constexpr int stage_n_threads = tile_n_size / 4;
  constexpr int stage_k_threads = has_perm ? tile_k_size : tile_ints;
  constexpr int stage_size = stage_k_threads * stage_n_threads;

  auto load_perm_to_shared = [&](int k_tile_id) {
    int first_k_int4 = (k_tile_id * tile_k_size) / 4;

    int4 const *perm_int4_ptr = reinterpret_cast<int4 const *>(perm_ptr);

    if (threadIdx.x < perm_size) {
      sh_perm_ptr[threadIdx.x] = perm_int4_ptr[first_k_int4 + threadIdx.x];
    }
    __syncthreads();
  };

  auto fetch_to_shared = [&](int pipe, int k_tile_id, int n_tile_id) {
    if (n_tile_id >= n_tiles) {
      cp_async_fence();
      return;
    }

    int first_n = n_tile_id * tile_n_size;

    int4 *sh_ptr = sh_pipe_ptr + stage_size * pipe;

    if constexpr (has_perm) {
      if (threadIdx.x < stage_size) {
        int k_id = threadIdx.x / stage_n_threads;
        int n_id = threadIdx.x % stage_n_threads;

        uint32_t const *sh_perm_int_ptr =
            reinterpret_cast<uint32_t const *>(sh_perm_ptr);

        int src_k = sh_perm_int_ptr[k_id];
        int src_k_packed = src_k / pack_factor;

        cp_async4(
            &sh_ptr[k_id * stage_n_threads + n_id],
            reinterpret_cast<int4 const *>(&(
                b_q_weight_ptr[src_k_packed * size_n + first_n + (n_id * 4)])));
      }

    } else {
      if (threadIdx.x < stage_size) {
        int k_id = threadIdx.x / stage_n_threads;
        int n_id = threadIdx.x % stage_n_threads;

        int first_k = k_tile_id * tile_k_size;
        int first_k_packed = first_k / pack_factor;

        cp_async4(&sh_ptr[k_id * stage_n_threads + n_id],
                  reinterpret_cast<int4 const *>(
                      &(b_q_weight_ptr[(first_k_packed + k_id) * size_n +
                                       first_n + (n_id * 4)])));
      }
    }

    cp_async_fence();
  };

  auto repack_tile = [&](int pipe, int k_tile_id, int n_tile_id) {
    if (n_tile_id >= n_tiles) {
      return;
    }

    int warp_id = threadIdx.x / 32;
    int th_id = threadIdx.x % 32;

    if (warp_id >= 4) {
      return;
    }

    int tc_col = th_id / 4;
    int tc_row = (th_id % 4) * 2;

    constexpr int tc_offsets[4] = {0, 1, 8, 9};

    int cur_n = warp_id * 16 + tc_col;

    constexpr int sh_stride = 64;
    constexpr uint32_t mask = (1 << num_bits) - 1;

    int4 *sh_stage_ptr = sh_pipe_ptr + stage_size * pipe;
    uint32_t *sh_stage_int_ptr = reinterpret_cast<uint32_t *>(sh_stage_ptr);

    uint32_t *sh_perm_int_ptr = reinterpret_cast<uint32_t *>(sh_perm_ptr);

    uint32_t vals[8];

    if constexpr (has_perm) {
      for (int i = 0; i < 4; i++) {
        int k_idx = tc_row + tc_offsets[i];

        uint32_t src_k = sh_perm_int_ptr[k_idx];
        uint32_t src_k_pos = src_k % pack_factor;

        uint32_t b1_val = sh_stage_int_ptr[k_idx * sh_stride + cur_n];
        uint32_t b1_cur_val = (b1_val >> (src_k_pos * num_bits)) & mask;

        uint32_t b2_val = sh_stage_int_ptr[k_idx * sh_stride + cur_n + 8];
        uint32_t b2_cur_val = (b2_val >> (src_k_pos * num_bits)) & mask;

        vals[i] = b1_cur_val;
        vals[4 + i] = b2_cur_val;
      }

    } else {
      uint32_t b1_vals[tile_ints];
      uint32_t b2_vals[tile_ints];

#pragma unroll
      for (int i = 0; i < tile_ints; i++) {
        b1_vals[i] = sh_stage_int_ptr[cur_n + sh_stride * i];
        b2_vals[i] = sh_stage_int_ptr[cur_n + 8 + sh_stride * i];
      }

#pragma unroll
      for (int i = 0; i < 4; i++) {
        int cur_elem = tc_row + tc_offsets[i];
        int cur_int = cur_elem / pack_factor;
        int cur_pos = cur_elem % pack_factor;

        vals[i] = (b1_vals[cur_int] >> (cur_pos * num_bits)) & mask;
        vals[4 + i] = (b2_vals[cur_int] >> (cur_pos * num_bits)) & mask;
      }
    }

    constexpr int tile_size = tile_k_size * tile_n_size / pack_factor;
    int out_offset = (k_tile_id * n_tiles + n_tile_id) * tile_size;

    // Result of:
    // https://github.com/NVIDIA/FasterTransformer/blob/main/src/fastertransformer/cutlass_extensions/include/cutlass_extensions/interleaved_numeric_conversion.h
    if constexpr (num_bits == 4) {
      constexpr int pack_idx[8] = {0, 2, 4, 6, 1, 3, 5, 7};

      uint32_t res = 0;
#pragma unroll
      for (int i = 0; i < 8; i++) {
        res |= vals[pack_idx[i]] << (i * 4);
      }

      out_ptr[out_offset + th_id * 4 + warp_id] = res;

    } else {
      constexpr int pack_idx[4] = {0, 2, 1, 3};

      uint32_t res1 = 0;
      uint32_t res2 = 0;
#pragma unroll
      for (int i = 0; i < 4; i++) {
        res1 |= vals[pack_idx[i]] << (i * 8);
        res2 |= vals[4 + pack_idx[i]] << (i * 8);
      }

      out_ptr[out_offset + th_id * 8 + (warp_id * 2) + 0] = res1;
      out_ptr[out_offset + th_id * 8 + (warp_id * 2) + 1] = res2;
    }
  };

  auto start_pipes = [&](int k_tile_id, int n_tile_id) {
#pragma unroll
    for (int pipe = 0; pipe < repack_stages - 1; pipe++) {
      fetch_to_shared(pipe, k_tile_id, n_tile_id + pipe);
    }

    wait_for_stage();
  };
#pragma unroll
  for (int k_tile_id = start_k_tile; k_tile_id < finish_k_tile; k_tile_id++) {
    int n_tile_id = 0;

    if constexpr (has_perm) {
      load_perm_to_shared(k_tile_id);
    }

    start_pipes(k_tile_id, n_tile_id);

    while (n_tile_id < n_tiles) {
#pragma unroll
      for (int pipe = 0; pipe < repack_stages; pipe++) {
        fetch_to_shared((pipe + repack_stages - 1) % repack_stages, k_tile_id,
                        n_tile_id + pipe + repack_stages - 1);
        repack_tile(pipe, k_tile_id, n_tile_id + pipe);
        wait_for_stage();
      }
      n_tile_id += repack_stages;
    }
  }
}

#define CALL_IF2(NUM_BITS, HAS_PERM)                                           \
  else if (num_bits == NUM_BITS && has_perm == HAS_PERM) {                     \
    hipFuncSetAttribute(reinterpret_cast<const void*>(                                                      \
        gptq_marlin_repack_kernel<repack_threads), NUM_BITS, HAS_PERM>,         \
        hipFuncAttributeMaxDynamicSharedMemorySize, max_shared_mem);          \
    gptq_marlin_repack_kernel<repack_threads, NUM_BITS, HAS_PERM>              \
        <<<blocks, repack_threads, max_shared_mem, stream>>>(                  \
            b_q_weight_ptr, perm_ptr, out_ptr, size_k, size_n);                \
  }

extern "C" void gptq_marlin_repack(void *weight, void *perm, void *out,
                                   int size_k, int size_n, int num_bits,
                                   int64_t stream_) {
  // Verify compatibility with marlin tile of 16x64
  assert(size_k % tile_k_size == 0);
  assert(size_n % tile_n_size == 0);
  assert(num_bits == 4 || num_bits == 8);
  const int pack_factor = 32 / num_bits;

  bool has_perm = true;
  int dev = 0;
  // Get ptrs
  uint32_t const *b_q_weight_ptr = reinterpret_cast<uint32_t const *>(weight);
  uint32_t const *perm_ptr = reinterpret_cast<uint32_t const *>(perm);
  uint32_t *out_ptr = reinterpret_cast<uint32_t *>(out);

  // Get dev info
  hipStream_t stream = (hipStream_t)stream_;
  int blocks;
  hipDeviceGetAttribute(&blocks, hipDeviceAttributeMultiprocessorCount, dev);

  int max_shared_mem = 0;
  hipDeviceGetAttribute(&max_shared_mem,
                         hipDeviceAttributeSharedMemPerBlockOptin, dev);
  assert(max_shared_mem > 0);

  if (false) {
  }
  CALL_IF2(4, false)
  CALL_IF2(4, true)
  CALL_IF2(8, false)
  CALL_IF2(8, true)
  else {
    assert(false);
  }
}

template <int const num_threads, int const num_bits>
__global__ void
awq_marlin_repack_kernel(uint32_t const *__restrict__ b_q_weight_ptr,
                         uint32_t *__restrict__ out_ptr, int size_k,
                         int size_n) {
  constexpr int pack_factor = 32 / num_bits;

  int k_tiles = size_k / tile_k_size;
  int n_tiles = size_n / tile_n_size;
  int block_k_tiles = div_ceil(k_tiles, gridDim.x);

  auto start_k_tile = blockIdx.x * block_k_tiles;
  if (start_k_tile >= k_tiles) {
    return;
  }

  int finish_k_tile = min(start_k_tile + block_k_tiles, k_tiles);

  // Wait until the next thread tile has been loaded to shared memory.
  auto wait_for_stage = [&]() {
    // We only have `stages - 2` active fetches since we are double buffering
    // and can only issue the next fetch when it is guaranteed that the previous
    // shared memory load is fully complete (as it may otherwise be
    // overwritten).
    cp_async_wait<repack_stages - 2>();
    __syncthreads();
  };

  extern __shared__ int4 sh[];

  constexpr int tile_n_ints = tile_n_size / pack_factor;

  constexpr int stage_n_threads = tile_n_ints / 4;
  constexpr int stage_k_threads = tile_k_size;
  constexpr int stage_size = stage_k_threads * stage_n_threads;

  auto fetch_to_shared = [&](int pipe, int k_tile_id, int n_tile_id) {
    if (n_tile_id >= n_tiles) {
      cp_async_fence();
      return;
    }

    int first_n = n_tile_id * tile_n_size;
    int first_n_packed = first_n / pack_factor;

    int4 *sh_ptr = sh + stage_size * pipe;

    if (threadIdx.x < stage_size) {
      auto k_id = threadIdx.x / stage_n_threads;
      auto n_id = threadIdx.x % stage_n_threads;

      int first_k = k_tile_id * tile_k_size;

      cp_async4(&sh_ptr[k_id * stage_n_threads + n_id],
                reinterpret_cast<int4 const *>(
                    &(b_q_weight_ptr[(first_k + k_id) * (size_n / pack_factor) +
                                     first_n_packed + (n_id * 4)])));
    }

    cp_async_fence();
  };

  auto repack_tile = [&](int pipe, int k_tile_id, int n_tile_id) {
    if (n_tile_id >= n_tiles) {
      return;
    }

    auto warp_id = threadIdx.x / 32;
    auto th_id = threadIdx.x % 32;

    if (warp_id >= 4) {
      return;
    }

    int tc_col = th_id / 4;
    int tc_row = (th_id % 4) * 2;

    constexpr int tc_offsets[4] = {0, 1, 8, 9};

    int cur_n = warp_id * 16 + tc_col;
    int cur_n_packed = cur_n / pack_factor;
    int cur_n_pos = cur_n % pack_factor;

    constexpr int sh_stride = tile_n_ints;
    constexpr uint32_t mask = (1 << num_bits) - 1;

    int4 *sh_stage_ptr = sh + stage_size * pipe;
    uint32_t *sh_stage_int_ptr = reinterpret_cast<uint32_t *>(sh_stage_ptr);

    // Undo interleaving
    int cur_n_pos_unpacked;
    if constexpr (num_bits == 4) {
      constexpr int undo_pack[8] = {0, 4, 1, 5, 2, 6, 3, 7};
      cur_n_pos_unpacked = undo_pack[cur_n_pos];
    } else {
      constexpr int undo_pack[4] = {0, 2, 1, 3};
      cur_n_pos_unpacked = undo_pack[cur_n_pos];
    }

    uint32_t vals[8];
#pragma unroll
    for (int i = 0; i < 4; i++) {
      int cur_elem = tc_row + tc_offsets[i];

      int packed_src_0 = sh_stage_int_ptr[cur_n_packed + sh_stride * cur_elem];
      int packed_src_1 = sh_stage_int_ptr[cur_n_packed + (8 / pack_factor) +
                                          sh_stride * cur_elem];

      vals[i] = (packed_src_0 >> (cur_n_pos_unpacked * num_bits)) & mask;
      vals[4 + i] = (packed_src_1 >> (cur_n_pos_unpacked * num_bits)) & mask;
    }

    constexpr int tile_size = tile_k_size * tile_n_size / pack_factor;
    int out_offset = (k_tile_id * n_tiles + n_tile_id) * tile_size;

    // Result of:
    // https://github.com/NVIDIA/FasterTransformer/blob/main/src/fastertransformer/cutlass_extensions/include/cutlass_extensions/interleaved_numeric_conversion.h
    if constexpr (num_bits == 4) {
      constexpr int pack_idx[8] = {0, 2, 4, 6, 1, 3, 5, 7};

      uint32_t res = 0;
#pragma unroll
      for (int i = 0; i < 8; i++) {
        res |= vals[pack_idx[i]] << (i * 4);
      }

      out_ptr[out_offset + th_id * 4 + warp_id] = res;

    } else {
      constexpr int pack_idx[4] = {0, 2, 1, 3};

      uint32_t res1 = 0;
      uint32_t res2 = 0;
#pragma unroll
      for (int i = 0; i < 4; i++) {
        res1 |= vals[pack_idx[i]] << (i * 8);
        res2 |= vals[4 + pack_idx[i]] << (i * 8);
      }

      out_ptr[out_offset + th_id * 8 + (warp_id * 2) + 0] = res1;
      out_ptr[out_offset + th_id * 8 + (warp_id * 2) + 1] = res2;
    }
  };

  auto start_pipes = [&](int k_tile_id, int n_tile_id) {
#pragma unroll
    for (int pipe = 0; pipe < repack_stages - 1; pipe++) {
      fetch_to_shared(pipe, k_tile_id, n_tile_id + pipe);
    }

    wait_for_stage();
  };
#pragma unroll
  for (int k_tile_id = start_k_tile; k_tile_id < finish_k_tile; k_tile_id++) {
    int n_tile_id = 0;

    start_pipes(k_tile_id, n_tile_id);

    while (n_tile_id < n_tiles) {
#pragma unroll
      for (int pipe = 0; pipe < repack_stages; pipe++) {
        fetch_to_shared((pipe + repack_stages - 1) % repack_stages, k_tile_id,
                        n_tile_id + pipe + repack_stages - 1);
        repack_tile(pipe, k_tile_id, n_tile_id + pipe);
        wait_for_stage();
      }
      n_tile_id += repack_stages;
    }
  }
}

#define CALL_IF3(NUM_BITS)                                                     \
  else if (num_bits == NUM_BITS) {                                             \
    hipFuncSetAttribute(reinterpret_cast<const void*>(awq_marlin_repack_kernel<repack_threads), NUM_BITS>,   \
                         hipFuncAttributeMaxDynamicSharedMemorySize,          \
                         max_shared_mem);                                      \
    awq_marlin_repack_kernel<repack_threads, NUM_BITS>                         \
        <<<blocks, repack_threads, max_shared_mem, stream>>>(                  \
            weight_ptr, out_ptr, size_k, size_n);                              \
  }

extern "C" void awq_marlin_repack(void *in, void *perm, void *out, int k, int n,
                                  int num_bits, int64_t stream_) {

  // in_dim 4096, out_dim 1024 (/pack_factor)
  // ws shape [4096, 128]
  // out_shape [256, 2048]

  // recover original size_k and size_n
  int const pack_factor = 32 / num_bits;
  int size_k = k;
  int size_n = n * pack_factor;

  // Verify compatibility with marlin tile of 16x64
  CHECK(size_k % marlin::tile_k_size == 0, "size_k = ", size_k,
        " is not divisible by tile_k_size = ", marlin::tile_k_size);
  CHECK(size_n % marlin::tile_n_size == 0, "size_n = ", size_n,
        " is not divisible by tile_n_size = ", marlin::tile_n_size);

  CHECK(num_bits == 4 || num_bits == 8,
        "num_bits must be 4 or 8. Got = ", num_bits);
  hipStream_t stream = (hipStream_t)stream_;

  uint32_t const *weight_ptr = reinterpret_cast<uint32_t const *>(in);
  uint32_t *out_ptr = reinterpret_cast<uint32_t *>(out);

  int blocks;
  hipDeviceGetAttribute(&blocks, hipDeviceAttributeMultiprocessorCount, 0);
  int max_shared_mem = 0;
  hipDeviceGetAttribute(&max_shared_mem,
                         hipDeviceAttributeSharedMemPerBlockOptin, 0);
  CHECK(max_shared_mem > 0, "error max_shared_mem");

  if (false) {
  }
  CALL_IF3(4)
  CALL_IF3(8)
  else {
    CHECK(false, "Unsupported repack config: num_bits = ", num_bits);
  }
}