#include "hip/hip_runtime.h"
/*
 * Copyright (C) Marlin.2024 Elias Frantar (elias.frantar@ist.ac.at)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#ifndef MARLIN_CUDA_KERNEL_CUH
#define MARLIN_CUDA_KERNEL_CUH
#include "marlin/marlin_dtypes.cuh"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <iostream>
using namespace marlin;

// m16n8k16 tensor core mma instruction with fp16/bf16 inputs and fp32
// output/accumulation.
template <typename scalar_t>
__device__ inline void mma(const typename ScalarType<scalar_t>::FragA &a_frag,
                           const typename ScalarType<scalar_t>::FragB &frag_b,
                           typename ScalarType<scalar_t>::FragC &frag_c) {
  const uint32_t *a = reinterpret_cast<const uint32_t *>(&a_frag);
  const uint32_t *b = reinterpret_cast<const uint32_t *>(&frag_b);
  float *c = reinterpret_cast<float *>(&frag_c);
  if constexpr (std::is_same<scalar_t, half>::value) {
    asm volatile("mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 "
                 "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};\n"
                 : "=f"(c[0]), "=f"(c[1]), "=f"(c[2]), "=f"(c[3])
                 : "r"(a[0]), "r"(a[1]), "r"(a[2]), "r"(a[3]), "r"(b[0]),
                   "r"(b[1]), "f"(c[0]), "f"(c[1]), "f"(c[2]), "f"(c[3]));
  } else if constexpr (std::is_same<scalar_t, hip_bfloat16>::value) {
    asm volatile("mma.sync.aligned.m16n8k16.row.col.f32.bf16.bf16.f32 "
                 "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};\n"
                 : "=f"(c[0]), "=f"(c[1]), "=f"(c[2]), "=f"(c[3])
                 : "r"(a[0]), "r"(a[1]), "r"(a[2]), "r"(a[3]), "r"(b[0]),
                   "r"(b[1]), "f"(c[0]), "f"(c[1]), "f"(c[2]), "f"(c[3]));
  }
}

// Instruction for loading a full 16x16 matrix fragment of operand A from shared
// memory, directly in tensor core layout.
template <typename scalar_t>
__device__ inline void ldsm4(typename ScalarType<scalar_t>::FragA &frag_a,
                             const void *smem_ptr) {
  uint32_t *a = reinterpret_cast<uint32_t *>(&frag_a);
  uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
  asm volatile("ldmatrix.sync.aligned.m8n8.x4.shared.b16 {%0,%1,%2,%3}, [%4];\n"
               : "=r"(a[0]), "=r"(a[1]), "=r"(a[2]), "=r"(a[3])
               : "r"(smem));
}

// Lookup-table based 3-input logical operation; explicitly used for
// dequantization as the compiler does not seem to automatically recognize it in
// all cases.
template <int lut> __device__ inline int lop3(int a, int b, int c) {
  int res;
  asm volatile("lop3.b32 %0, %1, %2, %3, %4;\n"
               : "=r"(res)
               : "r"(a), "r"(b), "r"(c), "n"(lut));
  return res;
}


template <typename scalar_t, ScalarTypeID w_type_id>
__device__ inline typename ScalarType<scalar_t>::FragB dequant(int q);

// Efficiently dequantize an int32 value into a full B-fragment of 4 fp16
// values. We mostly follow the strategy in the link below, with some small
// changes:
// https://github.com/NVIDIA/FasterTransformer/blob/main/src/fastertransformer/cutlass_extensions/include/cutlass_extensions/interleaved_numeric_conversion.h

//gptq dequant
template <>
__device__ inline typename ScalarType<half>::FragB
      dequant<half, ScalarTypeID::kU4B8>(int q) {
  const int LO = 0x000f000f;
  const int HI = 0x00f000f0;
  const int EX = 0x64006400;
  // Guarantee that the `(a & b) | c` operations are LOP3s.
  int lo = lop3 < (0xf0 & 0xcc) | 0xaa > (q, LO, EX);
  int hi = lop3 < (0xf0 & 0xcc) | 0xaa > (q, HI, EX);
  // We want signed int4 outputs, hence we fuse the `-8` symmetric zero point
  // directly into `SUB` and `ADD`.
  const int SUB = 0x64086408;
  const int MUL = 0x2c002c00;
  const int ADD = 0xd480d480;
  typename ScalarType<half>::FragB frag_b;
  frag_b[0] = __hsub2(*reinterpret_cast<half2 *>(&lo),
                      *reinterpret_cast<const half2 *>(&SUB));
  frag_b[1] = __hfma2(*reinterpret_cast<half2 *>(&hi),
                      *reinterpret_cast<const half2 *>(&MUL),
                      *reinterpret_cast<const half2 *>(&ADD));
  return frag_b;
}

template <>
__device__ inline typename ScalarType<hip_bfloat16>::FragB
dequant<hip_bfloat16, ScalarTypeID::kU4B8>(int q) {
  static constexpr uint32_t MASK = 0x000f000f;
  static constexpr uint32_t EX = 0x43004300;

  // Guarantee that the `(a & b) | c` operations are LOP3s.

  int lo = lop3 < (0xf0 & 0xcc) | 0xaa > (q, MASK, EX);
  q >>= 4;
  int hi = lop3 < (0xf0 & 0xcc) | 0xaa > (q, MASK, EX);

  typename ScalarType<hip_bfloat16>::FragB frag_b;
  static constexpr uint32_t MUL = 0x3F803F80;
  static constexpr uint32_t ADD = 0xC308C308;

  frag_b[0] = __hfma2(*reinterpret_cast<nv_bfloat162 *>(&lo),
                      *reinterpret_cast<const nv_bfloat162 *>(&MUL),
                      *reinterpret_cast<const nv_bfloat162 *>(&ADD));
  frag_b[1] = __hfma2(*reinterpret_cast<nv_bfloat162 *>(&hi),
                      *reinterpret_cast<const nv_bfloat162 *>(&MUL),
                      *reinterpret_cast<const nv_bfloat162 *>(&ADD));
  return frag_b;
}

//awq dequant
template <>
__device__ inline typename ScalarType<half>::FragB
dequant<half, ScalarTypeID::kU4>(int q) {
  const int LO = 0x000f000f;
  const int HI = 0x00f000f0;
  const int EX = 0x64006400;
  // Guarantee that the `(a & b) | c` operations are LOP3s.
  int lo = lop3 < (0xf0 & 0xcc) | 0xaa > (q, LO, EX);
  int hi = lop3 < (0xf0 & 0xcc) | 0xaa > (q, HI, EX);

  const int SUB = 0x64006400;
  const int MUL = 0x2c002c00;
  const int ADD = 0xd400d400;
  typename ScalarType<half>::FragB frag_b;
  frag_b[0] = __hsub2(*reinterpret_cast<half2*>(&lo),
                      *reinterpret_cast<const half2*>(&SUB));
  frag_b[1] = __hfma2(*reinterpret_cast<half2*>(&hi),
                      *reinterpret_cast<const half2*>(&MUL),
                      *reinterpret_cast<const half2*>(&ADD));
  return frag_b;
}

template <>
__device__ inline typename ScalarType<hip_bfloat16>::FragB
dequant<hip_bfloat16, ScalarTypeID::kU4>(int q) {
  static constexpr uint32_t MASK = 0x000f000f;
  static constexpr uint32_t EX = 0x43004300;

  // Guarantee that the `(a & b) | c` operations are LOP3s.

  int lo = lop3 < (0xf0 & 0xcc) | 0xaa > (q, MASK, EX);
  q >>= 4;
  int hi = lop3 < (0xf0 & 0xcc) | 0xaa > (q, MASK, EX);

  typename ScalarType<hip_bfloat16>::FragB frag_b;
  static constexpr uint32_t MUL = 0x3F803F80;
  static constexpr uint32_t ADD = 0xC300C300;

  frag_b[0] = __hfma2(*reinterpret_cast<nv_bfloat162*>(&lo),
                      *reinterpret_cast<const nv_bfloat162*>(&MUL),
                      *reinterpret_cast<const nv_bfloat162*>(&ADD));
  frag_b[1] = __hfma2(*reinterpret_cast<nv_bfloat162*>(&hi),
                      *reinterpret_cast<const nv_bfloat162*>(&MUL),
                      *reinterpret_cast<const nv_bfloat162*>(&ADD));
  return frag_b;
}
// Multiply dequantized values by the corresponding quantization scale; used
// only for grouped quantization.
template <typename scalar_t>
__device__ inline void scale(typename ScalarType<scalar_t>::FragB &frag_b,
                             typename ScalarType<scalar_t>::FragS &frag_s,
                             int i) {
  using scalar_t2 = typename ScalarType<scalar_t>::scalar_t2;
  scalar_t2 s =
      ScalarType<scalar_t>::num2num2(reinterpret_cast<scalar_t *>(&frag_s)[i]);
  frag_b[0] = __hmul2(frag_b[0], s);
  frag_b[1] = __hmul2(frag_b[1], s);
}


template <typename scalar_t>
__device__ inline void sub_zp(typename ScalarType<scalar_t>::FragB& frag_b,
                              typename ScalarType<scalar_t>::scalar_t2& frag_zp,
                              int i) {
  using scalar_t2 = typename ScalarType<scalar_t>::scalar_t2;
  scalar_t2 zp =
      ScalarType<scalar_t>::num2num2(reinterpret_cast<scalar_t*>(&frag_zp)[i]);
  frag_b[0] = __hsub2(frag_b[0], zp);
  frag_b[1] = __hsub2(frag_b[1], zp);
}

// Same as above, but for act_order (each K is multiplied individually)
template <typename scalar_t>
__device__ inline void scale4(typename ScalarType<scalar_t>::FragB& frag_b,
                              typename ScalarType<scalar_t>::FragS& frag_s_1,
                              typename ScalarType<scalar_t>::FragS& frag_s_2,
                              typename ScalarType<scalar_t>::FragS& frag_s_3,
                              typename ScalarType<scalar_t>::FragS& frag_s_4,
                              int i) {
  using scalar_t2 = typename ScalarType<scalar_t>::scalar_t2;
  scalar_t2 s_val_1_2;
  s_val_1_2.x = reinterpret_cast<scalar_t*>(&frag_s_1)[i];
  s_val_1_2.y = reinterpret_cast<scalar_t*>(&frag_s_2)[i];

  scalar_t2 s_val_3_4;
  s_val_3_4.x = reinterpret_cast<scalar_t*>(&frag_s_3)[i];
  s_val_3_4.y = reinterpret_cast<scalar_t*>(&frag_s_4)[i];

  frag_b[0] = __hmul2(frag_b[0], s_val_1_2);
  frag_b[1] = __hmul2(frag_b[1], s_val_3_4);
}

template <typename scalar_t,
          const int threads,          // number of threads in a threadblock
          const int thread_m_blocks,  // number of 16x16 blocks in the m
                                      // dimension (batchsize) of the
                                      // threadblock
          const int thread_n_blocks,  // same for n dimension (output)
          const int thread_k_blocks,  // same for k dimension (reduction)
          const int stages,  // number of stages for the async global->shared
                             // fetch pipeline
          const int group_blocks = -1,  // number of consecutive 16x16 blocks
          const ScalarTypeID w_type_id,
          const bool has_act_order,     // whether act_order is enabled
          const bool has_zp,            // whether zero-points are enabled
          const int num_bits
                                       // with a separate quantization scale
          >
__global__ void Marlin(
    const int4* __restrict__ A,  // fp16 input matrix of shape mxk
    const int4* __restrict__ B,  // 4bit quantized weight matrix of shape kxn
    int4* __restrict__ C,        // fp16 output buffer of shape mxn
    const int4* __restrict__ scales_ptr,  // fp16 quantization scales of shape
    const int4* __restrict__ zp_ptr,      // 4bit packed zero-points of shape
                                          // (k/groupsize)x(n/pack_factor)
    const int* __restrict__ g_idx,        // int32 group indices of shape k
                                 // (k/groupsize)xn
    int prob_m,                  // batch dimension m
    int prob_n,                  // output dimension n
    int prob_k,                  // reduction dimension k
    int num_groups,
    int* locks  // extra global storage for barrier synchronization
) {
  // Each threadblock processes one "stripe" of the B matrix with (roughly) the
  // same size, which might involve multiple column "slices" (of width 16 *
  // `thread_n_blocks`). Stripes are defined as shown in the 3x3 matrix 5 SM
  // example:
  //   0 1 3
  //   0 2 3
  //   1 2 4
  // While this kind of partitioning makes things somewhat more complicated, it
  // ensures good utilization of all SMs for many kinds of shape and GPU
  // configurations, while requiring as few slow global cross-threadblock
  // reductions as possible.
  using Dtype = ScalarType<scalar_t>;
  using scalar_t2 = typename ScalarType<scalar_t>::scalar_t2;
  using FragA = typename ScalarType<scalar_t>::FragA;
  using FragB = typename ScalarType<scalar_t>::FragB;
  using FragC = typename ScalarType<scalar_t>::FragC;
  using FragS = typename ScalarType<scalar_t>::FragS;
  using FragZP = typename ScalarType<scalar_t>::FragZP;

  const bool is_zp_float = false;
  // static constexpr auto w_type = vllm::ScalarType::from_id(w_type_id);

  constexpr int pack_factor = 32 / num_bits;

  // For larger GEMMs we run multiple batchsize 64 versions in parallel for a
  // better partitioning with less reductions
  int parallel = 1;
  if (prob_m > 16 * thread_m_blocks) {
    parallel = prob_m / (16 * thread_m_blocks);
    prob_m = 16 * thread_m_blocks;
  }

  int k_tiles = prob_k / 16 / thread_k_blocks;
  int n_tiles = prob_n / 16 / thread_n_blocks;
  int iters = div_ceil(k_tiles * n_tiles * parallel, gridDim.x);

  if constexpr (!has_act_order && group_blocks != -1) {
    if (group_blocks >= thread_k_blocks) {
      // Ensure that the number of tiles in each stripe is a multiple of the
      // groupsize; this avoids an annoying special case where a stripe starts
      // in the middle of group.
      iters = (group_blocks / thread_k_blocks) *
              div_ceil(iters, (group_blocks / thread_k_blocks));
    }
  }

  int slice_row = (iters * blockIdx.x) % k_tiles;
  int slice_col_par = (iters * blockIdx.x) / k_tiles;
  int slice_col = slice_col_par;
  int slice_iters;  // number of threadblock tiles in the current slice
  int slice_count =
      0;          // total number of active threadblocks in the current slice
  int slice_idx;  // index of threadblock in current slice; numbered bottom to
                  // top

  int par_id = 0;

  // We can easily implement parallel problem execution by just remapping
  // indices and advancing global pointers
  if (slice_col_par >= n_tiles) {
    A += (slice_col_par / n_tiles) * 16 * thread_m_blocks * prob_k / 8;
    C += (slice_col_par / n_tiles) * 16 * thread_m_blocks * prob_n / 8;
    locks += (slice_col_par / n_tiles) * n_tiles;
    slice_col = slice_col_par % n_tiles;
    par_id = slice_col_par / n_tiles;
  }

  // Compute all information about the current slice which is required for
  // synchronization.
  auto init_slice = [&]() {
    slice_iters =
        iters * (blockIdx.x + 1) - (k_tiles * slice_col_par + slice_row);
    if (slice_iters < 0 || slice_col_par >= n_tiles * parallel) slice_iters = 0;
    if (slice_iters == 0) return;
    if (slice_row + slice_iters > k_tiles) slice_iters = k_tiles - slice_row;
    slice_count = 1;
    slice_idx = 0;
    int col_first = iters * div_ceil(k_tiles * slice_col_par, iters);
    if (col_first <= k_tiles * (slice_col_par + 1)) {
      int col_off = col_first - k_tiles * slice_col_par;
      slice_count = div_ceil(k_tiles - col_off, iters);
      if (col_off > 0) slice_count++;
      int delta_first = iters * blockIdx.x - col_first;
      if (delta_first < 0 || (col_off == 0 && delta_first == 0))
        slice_idx = slice_count - 1;
      else {
        slice_idx = slice_count - 1 - delta_first / iters;
        if (col_off > 0) slice_idx--;
      }
    }
    if (slice_col == n_tiles) {
      A += 16 * thread_m_blocks * prob_k / 8;
      C += 16 * thread_m_blocks * prob_n / 8;
      locks += n_tiles;
      slice_col = 0;
      par_id++;
    }
  };
  init_slice();

  // A sizes/strides

  // stride of the A matrix in global memory
  int a_gl_stride = prob_k / 8;
  // stride of an A matrix tile in shared memory
  constexpr int a_sh_stride = 16 * thread_k_blocks / 8;
  // delta between subsequent A tiles in global memory
  constexpr int a_gl_rd_delta_o = 16 * thread_k_blocks / 8;
  // between subsequent accesses within a tile
  int a_gl_rd_delta_i = a_gl_stride * (threads / a_gl_rd_delta_o);
  // between shared memory writes
  constexpr int a_sh_wr_delta = a_sh_stride * (threads / a_gl_rd_delta_o);
  // between shared memory tile reads
  constexpr int a_sh_rd_delta_o = 2 * ((threads / 32) / (thread_n_blocks / 4));
  // within a shared memory tile
  constexpr int a_sh_rd_delta_i = a_sh_stride * 16;
  // overall size of a tile
  constexpr int a_sh_stage = a_sh_stride * (16 * thread_m_blocks);
  // number of shared write iterations for a tile
  constexpr int a_sh_wr_iters = div_ceil(a_sh_stage, a_sh_wr_delta);

  // B sizes/strides
  int b_gl_stride = 16 * prob_n / (pack_factor * 4);
  constexpr int b_sh_stride = ((thread_n_blocks * 16) * 16 / pack_factor) / 4;
  constexpr int b_thread_vecs = num_bits == 4 ? 1 : 2;
  constexpr int b_sh_stride_threads = b_sh_stride / b_thread_vecs;

  int b_gl_rd_delta_o = b_gl_stride * thread_k_blocks;
  int b_gl_rd_delta_i = b_gl_stride * (threads / b_sh_stride_threads);
  constexpr int b_sh_wr_delta = threads * b_thread_vecs;
  constexpr int b_sh_rd_delta = threads * b_thread_vecs;
  constexpr int b_sh_stage = b_sh_stride * thread_k_blocks;
  constexpr int b_sh_wr_iters = b_sh_stage / b_sh_wr_delta;

  // Scale sizes/strides without act_order
  int s_gl_stride = prob_n / 8;
  constexpr int s_sh_stride = 16 * thread_n_blocks / 8;
  constexpr int s_tb_groups =
      !has_act_order && group_blocks != -1 && group_blocks < thread_k_blocks
          ? thread_k_blocks / group_blocks
          : 1;
  constexpr int s_sh_stage = s_tb_groups * s_sh_stride;
  int s_gl_rd_delta = s_gl_stride;

  // Scale size/strides with act_order
  constexpr int tb_k = 16 * thread_k_blocks;
  constexpr int g_idx_stage = has_act_order ? (tb_k * sizeof(int)) / 16 : 0;
  // constexpr int act_s_row_stride      = 1;
  // int           act_s_col_stride      = act_s_row_stride * num_groups;
  int act_s_col_stride = 1;
  int act_s_col_warp_stride = act_s_col_stride * 8;
  int tb_n_warps = thread_n_blocks / 4;
  int act_s_col_tb_stride = act_s_col_warp_stride * tb_n_warps;

  // Zero-points sizes/strides
  int zp_gl_stride = is_zp_float ? prob_n / 8 : (prob_n / pack_factor) / 4;
  constexpr int zp_sh_stride = is_zp_float
                                   ? 16 * thread_n_blocks / 8
                                   : ((16 * thread_n_blocks) / pack_factor) / 4;
  constexpr int zp_tb_groups = s_tb_groups;
  constexpr int zp_sh_stage = has_zp ? zp_tb_groups * zp_sh_stride : 0;
  int zp_gl_rd_delta = zp_gl_stride;

  // Global A read index of current thread.
  int a_gl_rd = a_gl_stride * (threadIdx.x / a_gl_rd_delta_o) +
                (threadIdx.x % a_gl_rd_delta_o);
  a_gl_rd += a_gl_rd_delta_o * slice_row;
  // Shared write index of current thread.
  int a_sh_wr = a_sh_stride * (threadIdx.x / a_gl_rd_delta_o) +
                (threadIdx.x % a_gl_rd_delta_o);
  // Shared read index.
  int a_sh_rd =
      a_sh_stride * ((threadIdx.x % 32) % 16) + (threadIdx.x % 32) / 16;
  a_sh_rd += 2 * ((threadIdx.x / 32) / (thread_n_blocks / 4));

  int b_gl_rd = b_gl_stride * (threadIdx.x / b_sh_stride_threads) +
                (threadIdx.x % b_sh_stride_threads) * b_thread_vecs;
  b_gl_rd += b_sh_stride * slice_col;
  b_gl_rd += b_gl_rd_delta_o * slice_row;
  auto b_sh_wr = threadIdx.x * b_thread_vecs;
  auto b_sh_rd = threadIdx.x * b_thread_vecs;

  // For act_order
  constexpr int k_iter_size = tb_k / b_sh_wr_iters;
  int slice_k_start = tb_k * slice_row;
  int slice_k_finish = slice_k_start + tb_k * slice_iters;
  int slice_k_start_shared_fetch = slice_k_start;
  int slice_n_offset = act_s_col_tb_stride * slice_col;

  // No act_order
  int s_gl_rd;
  if constexpr (!has_act_order) {
    if constexpr (group_blocks == -1) {
      s_gl_rd = s_sh_stride * slice_col + threadIdx.x;
    } else {
      s_gl_rd = s_gl_stride * ((thread_k_blocks * slice_row) / group_blocks) +
                s_sh_stride * slice_col + threadIdx.x;
    }
  }
  auto s_sh_wr = threadIdx.x;
  bool s_sh_wr_pred = threadIdx.x < s_sh_stride;

  // Zero-points
  int zp_gl_rd;
  if constexpr (has_zp) {
    if constexpr (group_blocks == -1) {
      zp_gl_rd = zp_sh_stride * slice_col + threadIdx.x;
    } else {
      zp_gl_rd = zp_gl_stride * ((thread_k_blocks * slice_row) / group_blocks) +
                 zp_sh_stride * slice_col + threadIdx.x;
    }
  }
  auto zp_sh_wr = threadIdx.x;
  bool zp_sh_wr_pred = threadIdx.x < zp_sh_stride;

  // We use a different scale layout for grouped and column-wise quantization as
  // we scale a `half2` tile in column-major layout in the former and in
  // row-major in the latter case.
  int s_sh_rd;
  if constexpr (group_blocks != -1)
    s_sh_rd = 8 * ((threadIdx.x / 32) % (thread_n_blocks / 4)) +
              (threadIdx.x % 32) / 4;
  else
    s_sh_rd = 8 * ((threadIdx.x / 32) % (thread_n_blocks / 4)) +
              (threadIdx.x % 32) % 4;

  // Zero-points have the same read layout as the scales
  // (without column-wise case)
  constexpr int num_col_threads = 8;
  constexpr int num_row_threads = 4;
  constexpr int num_ints_per_thread = 8 / pack_factor;
  int zp_sh_rd;
  if constexpr (has_zp) {
    if constexpr (is_zp_float) {
      if constexpr (group_blocks != -1) {
        zp_sh_rd = 8 * ((threadIdx.x / 32) % (thread_n_blocks / 4)) +
                   (threadIdx.x % 32) / 4;
      }
    } else {
      zp_sh_rd = num_ints_per_thread * num_col_threads *
                     ((threadIdx.x / 32) % (thread_n_blocks / 4)) +
                 num_ints_per_thread * ((threadIdx.x % 32) / num_row_threads);
    }
  }

  // Precompute which thread should not read memory in which iterations; this is
  // needed if there are more threads than required for a certain tilesize or
  // when the batchsize is not a multiple of 16.
  bool a_sh_wr_pred[a_sh_wr_iters];
  #pragma unroll
  for (int i = 0; i < a_sh_wr_iters; i++)
    a_sh_wr_pred[i] = a_sh_wr_delta * i + a_sh_wr < a_sh_stride * prob_m;

  // To ensure that writing and reading A tiles to/from shared memory, the
  // latter in fragment format, is fully bank conflict free, we need to use a
  // rather fancy XOR-based layout. The key here is that neither reads nor
  // writes of the 16-byte `int4` blocks of 8 consecutive threads involve the
  // same shared memory banks. Further, it seems (based on NSight-Compute) that
  // each warp must also write a consecutive memory segment?
  auto transform_a = [&](int i) {
    int row = i / a_gl_rd_delta_o;
    return a_gl_rd_delta_o * row + (i % a_gl_rd_delta_o) ^ row;
  };
  // Since the computation of this remapping is non-trivial and, due to our main
  // loop unrolls, all shared memory accesses are static, we simply precompute
  // both transformed reads and writes.
  int a_sh_wr_trans[a_sh_wr_iters];
  #pragma unroll
  for (int i = 0; i < a_sh_wr_iters; i++)
    a_sh_wr_trans[i] = transform_a(a_sh_wr_delta * i + a_sh_wr);
  int a_sh_rd_trans[b_sh_wr_iters][thread_m_blocks];
  #pragma unroll
  for (int i = 0; i < b_sh_wr_iters; i++) {
  #pragma unroll
    for (int j = 0; j < thread_m_blocks; j++)
      a_sh_rd_trans[i][j] =
          transform_a(a_sh_rd_delta_o * i + a_sh_rd_delta_i * j + a_sh_rd);
  }

  // Since B-accesses have non-constant stride they have to be computed at
  // runtime; we break dependencies between subsequent accesses with a tile by
  // maintining multiple pointers (we have enough registers), a tiny
  // optimization.
  const int4* B_ptr[b_sh_wr_iters];
  #pragma unroll
  for (int i = 0; i < b_sh_wr_iters; i++)
    B_ptr[i] = B + b_gl_rd_delta_i * i + b_gl_rd;

  extern __shared__ int4 sh[];
  // Shared memory storage for global fetch pipelines.
  int4* sh_a = sh;
  int4* sh_b = sh_a + (stages * a_sh_stage);
  int4* sh_g_idx = sh_b + (stages * b_sh_stage);
  int4* sh_zp = sh_g_idx + (stages * g_idx_stage);
  int4* sh_s = sh_zp + (stages * zp_sh_stage);
  int4* sh_red = sh_s + (stages * s_sh_stage);

  // Register storage for double buffer of shared memory reads.
  FragA frag_a[2][thread_m_blocks];
  I4 frag_b_quant[2][b_thread_vecs];
  FragC frag_c[thread_m_blocks][4][2];
  FragS frag_s[2][4];                    // No act-order
  FragS act_frag_s[2][4][4];             // For act-order
  int frag_qzp[2][num_ints_per_thread];  // Zero-points
  FragZP frag_zp;                        // Zero-points in fp16
  FragZP frag_zpf[2];                    // Zero-points in fp16 in HQQ

  // Zero accumulators.
  auto zero_accums = [&]() {
  #pragma unroll
    for (int i = 0; i < thread_m_blocks * 4 * 2 * 4; i++)
      reinterpret_cast<float*>(frag_c)[i] = 0;
  };

  int sh_first_group_id = -1;
  int sh_num_groups = -1;
  constexpr int sh_max_num_groups = 32;

  auto fetch_scales_to_shared = [&](bool is_async, int first_group_id,
                                    int last_group_id) {
    sh_first_group_id = first_group_id;
    sh_num_groups = last_group_id - first_group_id + 1;

    if (sh_num_groups < sh_max_num_groups) {
      sh_num_groups = sh_max_num_groups;
    }

    if (sh_first_group_id + sh_num_groups > num_groups) {
      sh_num_groups = num_groups - sh_first_group_id;
    }

    int row_offset = first_group_id * s_gl_stride;

    if (is_async) {
      for (int i = 0; i < sh_num_groups; i++) {
        if (threadIdx.x < s_sh_stride) {
          cp_async4_pred(&sh_s[(i * s_sh_stride) + threadIdx.x],
                         &scales_ptr[row_offset + (i * s_gl_stride) +
                                     slice_n_offset + threadIdx.x]);
        }
      }
    } else {
      for (int i = 0; i < sh_num_groups; i++) {
        if (threadIdx.x < s_sh_stride) {
          sh_s[(i * s_sh_stride) + threadIdx.x] =
              scales_ptr[row_offset + (i * s_gl_stride) + slice_n_offset +
                         threadIdx.x];
        }
      }
    }
  };
  // Asynchronously fetch the next A, B and s tile from global to the next
  // shared memory pipeline location.
  auto fetch_to_shared = [&](int pipe, int a_off, bool pred = true) {
    if (pred) {
      int4* sh_a_stage = sh_a + a_sh_stage * pipe;
  #pragma unroll
      for (int i = 0; i < a_sh_wr_iters; i++) {
        cp_async4_pred(
            &sh_a_stage[a_sh_wr_trans[i]],
            &A[a_gl_rd_delta_i * i + a_gl_rd + a_gl_rd_delta_o * a_off],
            a_sh_wr_pred[i]);
      }
      int4* sh_b_stage = sh_b + b_sh_stage * pipe;
  #pragma unroll
      for (int i = 0; i < b_sh_wr_iters; i++) {
  #pragma unroll
        for (int j = 0; j < b_thread_vecs; j++) {
          cp_async4(&sh_b_stage[b_sh_wr_delta * i + b_sh_wr + j], B_ptr[i] + j);
        }

        B_ptr[i] += b_gl_rd_delta_o;
      }

      if constexpr (has_act_order) {
        // Fetch g_idx thread-block portion
        int full_pipe = a_off;
        int cur_k = slice_k_start_shared_fetch + tb_k * full_pipe;
        if (cur_k < prob_k && cur_k < slice_k_finish) {
          int4* sh_g_idx_stage = sh_g_idx + g_idx_stage * pipe;

          int4 const* cur_g_idx_stage_ptr =
              reinterpret_cast<int4 const*>(&g_idx[cur_k]);

          if (threadIdx.x < g_idx_stage) {
            cp_async4_pred(&sh_g_idx_stage[threadIdx.x],
                           &cur_g_idx_stage_ptr[threadIdx.x]);
          }
        }
      } else {
        if constexpr (group_blocks != -1) {
          int4* sh_s_stage = sh_s + s_sh_stage * pipe;

          if constexpr (group_blocks >= thread_k_blocks) {
            if (s_sh_wr_pred) {
              cp_async4(&sh_s_stage[s_sh_wr], &scales_ptr[s_gl_rd]);
            }
            // Only fetch scales if this tile starts a new group
            if ((pipe + 1) % (group_blocks / thread_k_blocks) == 0) {
              s_gl_rd += s_gl_rd_delta;
            }
          } else {
            for (int i = 0; i < s_tb_groups; i++) {
              if (s_sh_wr_pred) {
                cp_async4(&sh_s_stage[i * s_sh_stride + s_sh_wr],
                          &scales_ptr[s_gl_rd]);
              }
              s_gl_rd += s_gl_rd_delta;
            }
          }
        }

        if constexpr (has_zp && group_blocks != -1) {
          int4* sh_zp_stage = sh_zp + zp_sh_stage * pipe;

          if constexpr (group_blocks >= thread_k_blocks) {
            // Only fetch zero-points if this tile starts a new group
            if (pipe % (group_blocks / thread_k_blocks) == 0) {
              if (zp_sh_wr_pred) {
                cp_async4(&sh_zp_stage[zp_sh_wr], &zp_ptr[zp_gl_rd]);
              }
              zp_gl_rd += zp_gl_rd_delta;
            }
          } else {
            for (int i = 0; i < zp_tb_groups; i++) {
              if (zp_sh_wr_pred) {
                cp_async4(&sh_zp_stage[i * zp_sh_stride + zp_sh_wr],
                          &zp_ptr[zp_gl_rd]);
              }
              zp_gl_rd += zp_gl_rd_delta;
            }
          }
        }
      }
    }
    // Insert a fence even when we are winding down the pipeline to ensure that
    // waiting is also correct at this point.
    cp_async_fence();
  };

  auto fetch_zp_to_shared = [&]() {
    if (zp_sh_wr_pred) {
      cp_async4(&sh_zp[zp_sh_wr], &zp_ptr[zp_gl_rd]);
    }
  };

  // Wait until the next thread tile has been loaded to shared memory.
  auto wait_for_stage = [&]() {
    // We only have `stages - 2` active fetches since we are double buffering
    // and can only issue the next fetch when it is guaranteed that the previous
    // shared memory load is fully complete (as it may otherwise be
    // overwritten).
    cp_async_wait<stages - 2>();
    __syncthreads();
  };

  // Load the next sub-tile from the current location in the shared memory pipe
  // into the current register buffer.
  auto fetch_to_registers = [&](int k, int pipe) {
    int4* sh_a_stage = sh_a + a_sh_stage * pipe;
  #pragma unroll
    for (int i = 0; i < thread_m_blocks; i++)
      ldsm4<scalar_t>(frag_a[k % 2][i],
                      &sh_a_stage[a_sh_rd_trans[k % b_sh_wr_iters][i]]);
    int4* sh_b_stage = sh_b + b_sh_stage * pipe;

  #pragma unroll
    for (int i = 0; i < b_thread_vecs; i++) {
      frag_b_quant[k % 2][i] = *reinterpret_cast<I4*>(
          &sh_b_stage[b_sh_rd_delta * (k % b_sh_wr_iters) + b_sh_rd + i]);
    }
  };

  bool is_same_group[stages];
  int same_group_id[stages];

  auto init_same_group = [&](int pipe) {
    if constexpr (!has_act_order) {
      is_same_group[pipe] = false;
      same_group_id[pipe] = 0;
      return;
    }

    int4* sh_g_idx_stage = sh_g_idx + g_idx_stage * pipe;
    int* sh_g_idx_int_ptr = reinterpret_cast<int*>(sh_g_idx_stage);

    int group_id_1 = sh_g_idx_int_ptr[0];
    int group_id_2 = sh_g_idx_int_ptr[tb_k - 1];

    is_same_group[pipe] = group_id_1 == group_id_2;
    same_group_id[pipe] = group_id_1;
  };

  auto fetch_scales_to_registers = [&](int k, int full_pipe) {
    int pipe = full_pipe % stages;

    if constexpr (!has_act_order) {
      // No act-order case
      if constexpr (group_blocks != -1) {
        if constexpr (group_blocks >= thread_k_blocks) {
          int4* sh_s_stage = sh_s + s_sh_stage * pipe;
          reinterpret_cast<int4*>(&frag_s[k % 2])[0] = sh_s_stage[s_sh_rd];
        } else {
          auto warp_id = threadIdx.x / 32;
          int n_warps = thread_n_blocks / 4;

          int warp_row = warp_id / n_warps;

          int cur_k = warp_row * 16;
          cur_k += k_iter_size * (k % b_sh_wr_iters);

          int k_blocks = cur_k / 16;
          int cur_group_id = k_blocks / group_blocks;

          int4* sh_s_stage = sh_s + s_sh_stage * pipe;

          reinterpret_cast<int4*>(&frag_s[k % 2])[0] =
              sh_s_stage[s_sh_rd + cur_group_id * s_sh_stride];
        }
      }

      return;
    }

    // Act-order case

    // Determine K of the "current" thread-block
    int cur_k = slice_k_start + tb_k * full_pipe;
    if (cur_k >= prob_k || cur_k >= slice_k_finish) {
      return;
    }

    // Reset (to current thread-block) since we read g_idx portion from the
    // shared memory
    cur_k = 0;

    // Progress to current iteration
    cur_k += k_iter_size * (k % b_sh_wr_iters);

    // Determine "position" inside the thread-block (based on warp and
    // thread-id)
    auto warp_id = threadIdx.x / 32;
    int n_warps =
        thread_n_blocks / 4;  // Each warp processes 4 16-size tiles over N

    int warp_row = warp_id / n_warps;
    int warp_col = warp_id % n_warps;

    cur_k += warp_row * 16;

    auto th_id = threadIdx.x % 32;
    cur_k += (th_id % 4) * 2;  // Due to tensor-core layout for fp16 B matrix

    int s_col_shift =
        /*slice_n_offset +*/ (act_s_col_warp_stride * warp_col) +
        (th_id / 4) * act_s_col_stride;

    if (is_same_group[pipe]) {
      if (k % 2 == 0) {
        *(reinterpret_cast<int4*>(&(act_frag_s[k % 2][0][0]))) =
            sh_s[(same_group_id[pipe] - sh_first_group_id) * s_sh_stride +
                 s_col_shift];
      } else {
        *(reinterpret_cast<int4*>(&(act_frag_s[k % 2][0][0]))) =
            *(reinterpret_cast<int4*>(&(act_frag_s[(k - 1) % 2][0][0])));
      }

      for (int i = 1; i < 4; i++) {
        *(reinterpret_cast<int4*>(&(act_frag_s[k % 2][i][0]))) =
            *(reinterpret_cast<int4*>(&(act_frag_s[k % 2][0][0])));
      }
      return;
    }

    int4* sh_g_idx_stage = sh_g_idx + g_idx_stage * pipe;
    int* sh_g_idx_int_ptr = reinterpret_cast<int*>(sh_g_idx_stage);

    constexpr int k_frag_offsets[4] = {0, 1, 8,
                                       9};  // Tensor core offsets per thread

  #pragma unroll
    for (int i = 0; i < 4; i++) {
      int actual_k = cur_k + k_frag_offsets[i];

      int group_id = sh_g_idx_int_ptr[actual_k];
      int rel_group_id = group_id - sh_first_group_id;

      *(reinterpret_cast<int4*>(&(act_frag_s[k % 2][i][0]))) =
          sh_s[rel_group_id * s_sh_stride + s_col_shift];
    }
  };

  auto fetch_zp_to_registers = [&](int k, int full_pipe) {
    // This code does not handle group_blocks == 0,
    // which signifies act_order.
    // has_zp implies AWQ, which doesn't have act_order,
    static_assert(!has_zp || group_blocks != 0);

    if constexpr (has_zp) {
      int pipe = full_pipe % stages;

      if constexpr (group_blocks == -1) {
        for (int i = 0; i < num_ints_per_thread; i++) {
          frag_qzp[k % 2][i] = (reinterpret_cast<int*>(sh_zp))[zp_sh_rd + i];
        }

      } else if constexpr (group_blocks >= thread_k_blocks) {
        int4* sh_zp_stage =
            sh_zp + zp_sh_stage * ((group_blocks / thread_k_blocks) *
                                   (pipe / (group_blocks / thread_k_blocks)));
        for (int i = 0; i < num_ints_per_thread; i++) {
          frag_qzp[k % 2][i] =
              (reinterpret_cast<int*>(sh_zp_stage))[zp_sh_rd + i];
        }
      } else {
        auto warp_id = threadIdx.x / 32;
        int n_warps = thread_n_blocks / 4;

        int warp_row = warp_id / n_warps;

        int cur_k = warp_row * 16;
        cur_k += k_iter_size * (k % b_sh_wr_iters);

        int k_blocks = cur_k / 16;
        int cur_group_id = 0;

        // Suppress bogus and persistent divide-by-zero warning
  #pragma nv_diagnostic push
  #pragma nv_diag_suppress divide_by_zero
        cur_group_id = k_blocks / group_blocks;
  #pragma nv_diagnostic pop

        int4* sh_zp_stage = sh_zp + zp_sh_stage * pipe;

        sh_zp_stage += cur_group_id * zp_sh_stride;

        for (int i = 0; i < num_ints_per_thread; i++) {
          frag_qzp[k % 2][i] =
              (reinterpret_cast<int*>(sh_zp_stage))[zp_sh_rd + i];
        }
      }
    }
  };

  // Execute the actual tensor core matmul of a sub-tile.
  auto matmul = [&](int k) {
    if constexpr (has_zp) {
      FragB frag_zp_0;
      FragB frag_zp_1;
      int zp_quant_0, zp_quant_1;

      if constexpr (num_bits == 4) {
        zp_quant_0 = frag_qzp[k % 2][0];
        zp_quant_1 = zp_quant_0 >> 8;
      } else {
        static_assert(num_bits == 8);
        zp_quant_0 = frag_qzp[k % 2][0];
        zp_quant_1 = frag_qzp[k % 2][1];
      }

      frag_zp_0 = dequant<scalar_t, w_type_id>(zp_quant_0);
      frag_zp_1 = dequant<scalar_t, w_type_id>(zp_quant_1);

      frag_zp[0] = frag_zp_0[0];
      frag_zp[1] = frag_zp_0[1];
      frag_zp[2] = frag_zp_1[0];
      frag_zp[3] = frag_zp_1[1];
    }

  // We have the m dimension as the inner loop in order to encourage overlapping
  // dequantization and matmul operations.
  #pragma unroll
    for (int j = 0; j < 4; j++) {
      FragB frag_b0;
      FragB frag_b1;
      int b_quant_0, b_quant_1;

      if constexpr (num_bits == 4) {
        b_quant_0 = frag_b_quant[k % 2][0][j];
        b_quant_1 = b_quant_0 >> 8;
      }

      frag_b0 = dequant<scalar_t, w_type_id>(b_quant_0);
      frag_b1 = dequant<scalar_t, w_type_id>(b_quant_1);

      // Apply zero-point to frag_b0
      if constexpr (has_zp) {
        sub_zp<scalar_t>(frag_b0, frag_zp[j], 0);
      }

      // Apply scale to frag_b0
      if constexpr (has_act_order) {
        scale4<scalar_t>(frag_b0, act_frag_s[k % 2][0][j],
                         act_frag_s[k % 2][1][j], act_frag_s[k % 2][2][j],
                         act_frag_s[k % 2][3][j], 0);
      } else {
        if constexpr (group_blocks != -1) {
          scale<scalar_t>(frag_b0, frag_s[k % 2][j], 0);
        }
      }

      // Apply zero-point to frag_b1
      if constexpr (has_zp) {
        sub_zp<scalar_t>(frag_b1, frag_zp[j], 1);
      }

      // Apply scale to frag_b1
      if constexpr (has_act_order) {
        scale4<scalar_t>(frag_b1, act_frag_s[k % 2][0][j],
                         act_frag_s[k % 2][1][j], act_frag_s[k % 2][2][j],
                         act_frag_s[k % 2][3][j], 1);

      } else {
        if constexpr (group_blocks != -1) {
          scale<scalar_t>(frag_b1, frag_s[k % 2][j], 1);
        }
      }

  #pragma unroll
      for (int i = 0; i < thread_m_blocks; i++) {
        mma<scalar_t>(frag_a[k % 2][i], frag_b0, frag_c[i][j][0]);
        mma<scalar_t>(frag_a[k % 2][i], frag_b1, frag_c[i][j][1]);
      }
    }
  };

  // Since we slice across the k dimension of a tile in order to increase the
  // number of warps while keeping the n dimension of a tile reasonable, we have
  // multiple warps that accumulate their partial sums of the same output
  // location; which we have to reduce over in the end. We do in shared memory.
  auto thread_block_reduce = [&]() {
    constexpr int red_off = threads / b_sh_stride_threads / 2;
    if (red_off >= 1) {
      auto red_idx = threadIdx.x / b_sh_stride_threads;
      constexpr int red_sh_stride = b_sh_stride_threads * 4 * 2;
      constexpr int red_sh_delta = b_sh_stride_threads;
      int red_sh_rd = red_sh_stride * (threadIdx.x / b_sh_stride_threads) +
                      (threadIdx.x % b_sh_stride_threads);

      // Parallel logarithmic shared memory reduction. We make sure to avoid any
      // unnecessary read or write iterations, e.g., for two warps we write only
      // once by warp 1 and read only once by warp 0.

  #pragma unroll
      for (int m_block = 0; m_block < thread_m_blocks; m_block++) {
  #pragma unroll
        for (int i = red_off; i > 0; i /= 2) {
          if (i <= red_idx && red_idx < 2 * i) {
  #pragma unroll
            for (int j = 0; j < 4 * 2; j++) {
              int red_sh_wr =
                  red_sh_delta * j + (red_sh_rd - red_sh_stride * i);
              if (i < red_off) {
                float* c_rd = reinterpret_cast<float*>(
                    &sh_red[red_sh_delta * j + red_sh_rd]);
                float* c_wr = reinterpret_cast<float*>(&sh_red[red_sh_wr]);
  #pragma unroll
                for (int k = 0; k < 4; k++)
                  reinterpret_cast<FragC*>(frag_c)[4 * 2 * m_block + j][k] +=
                      c_rd[k] + c_wr[k];
              }
              sh_red[red_sh_wr] =
                  reinterpret_cast<int4*>(&frag_c)[4 * 2 * m_block + j];
            }
          }
          __syncthreads();
        }
        if (red_idx == 0) {
  #pragma unroll
          for (int i = 0; i < 4 * 2; i++) {
            float* c_rd =
                reinterpret_cast<float*>(&sh_red[red_sh_delta * i + red_sh_rd]);
  #pragma unroll
            for (int j = 0; j < 4; j++)
              reinterpret_cast<FragC*>(frag_c)[4 * 2 * m_block + i][j] +=
                  c_rd[j];
          }
        }
        __syncthreads();
      }
    }
  };

  // Since multiple threadblocks may process parts of the same column slice, we
  // finally have to globally reduce over the results. As the striped
  // partitioning minimizes the number of such reductions and our outputs are
  // usually rather small, we perform this reduction serially in L2 cache.
  auto global_reduce = [&](bool first = false, bool last = false) {
    // We are very careful here to reduce directly in the output buffer to
    // maximize L2 cache utilization in this step. To do this, we write out
    // results in FP16 (but still reduce with FP32 compute).
    constexpr int active_threads = 32 * thread_n_blocks / 4;
    if (threadIdx.x < active_threads) {
      int c_gl_stride = prob_n / 8;
      int c_gl_wr_delta_o = 8 * c_gl_stride;
      int c_gl_wr_delta_i = 4 * (active_threads / 32);
      int c_gl_wr = c_gl_stride * ((threadIdx.x % 32) / 4) +
                    4 * (threadIdx.x / 32) + threadIdx.x % 4;
      c_gl_wr += (2 * thread_n_blocks) * slice_col;
      constexpr int c_sh_wr_delta = active_threads;
      auto c_sh_wr = threadIdx.x;

      int row = (threadIdx.x % 32) / 4;

      if (!first) {
  // Interestingly, doing direct global accesses here really seems to mess up
  // the compiler and lead to slowdowns, hence we also use async-copies even
  // though these fetches are not actually asynchronous.
  #pragma unroll
        for (int i = 0; i < thread_m_blocks * 4; i++) {
          cp_async4_pred(
              &sh_red[c_sh_wr + c_sh_wr_delta * i],
              &C[c_gl_wr + c_gl_wr_delta_o * (i / 2) +
                 c_gl_wr_delta_i * (i % 2)],
              i < (thread_m_blocks - 1) * 4 || 8 * (i / 2) + row < prob_m);
        }
        cp_async_fence();
        cp_async_wait<0>();
      }

  #pragma unroll
      for (int i = 0; i < thread_m_blocks * 4; i++) {
        if (i < (thread_m_blocks - 1) * 4 || 8 * (i / 2) + row < prob_m) {
          if (!first) {
            int4 c_red = sh_red[c_sh_wr + i * c_sh_wr_delta];
  #pragma unroll
            for (int j = 0; j < 2 * 4; j++) {
              reinterpret_cast<float*>(
                  &frag_c)[4 * 2 * 4 * (i / 4) + 4 * j + (i % 4)] +=
                  Dtype::num2float(reinterpret_cast<scalar_t*>(&c_red)[j]);
            }
          }
          if (!last) {
            int4 c;
  #pragma unroll
            for (int j = 0; j < 2 * 4; j++) {
              reinterpret_cast<scalar_t*>(&c)[j] =
                  Dtype::float2num(reinterpret_cast<float*>(
                      &frag_c)[4 * 2 * 4 * (i / 4) + 4 * j + (i % 4)]);
            }
            C[c_gl_wr + c_gl_wr_delta_o * (i / 2) + c_gl_wr_delta_i * (i % 2)] =
                c;
          }
        }
      }
    }
  };

  // Write out the reduce final result in the correct layout. We only actually
  // reshuffle matrix fragments in this step, the reduction above is performed
  // in fragment layout.
  auto write_result = [&]() {
    int c_gl_stride = prob_n / 8;
    constexpr int c_sh_stride = 2 * thread_n_blocks + 1;
    int c_gl_wr_delta = c_gl_stride * (threads / (2 * thread_n_blocks));
    constexpr int c_sh_rd_delta =
        c_sh_stride * (threads / (2 * thread_n_blocks));

    int c_gl_wr = c_gl_stride * (threadIdx.x / (2 * thread_n_blocks)) +
                  (threadIdx.x % (2 * thread_n_blocks));
    c_gl_wr += (2 * thread_n_blocks) * slice_col;
    int c_sh_wr =
        (4 * c_sh_stride) * ((threadIdx.x % 32) / 4) + (threadIdx.x % 32) % 4;
    c_sh_wr += 32 * (threadIdx.x / 32);
    int c_sh_rd = c_sh_stride * (threadIdx.x / (2 * thread_n_blocks)) +
                  (threadIdx.x % (2 * thread_n_blocks));

    int c_gl_wr_end = c_gl_stride * prob_m;

    // We first reorder in shared memory to guarantee the most efficient final
    // global write patterns
    auto write = [&](int idx, float c0, float c1, FragS& s) {
      scalar_t2 res =
          Dtype::nums2num2(Dtype::float2num(c0), Dtype::float2num(c1));

      // For per-column quantization we finally apply the scale here (only for
      // 4-bit)
      if constexpr (!has_act_order && group_blocks == -1 &&
                    num_bits == 4) {
        res = __hmul2(res, s[0]);
      }

      ((scalar_t2*)sh_red)[idx] = res;
    };

    if (threadIdx.x / 32 < thread_n_blocks / 4) {
  #pragma unroll
      for (int i = 0; i < thread_m_blocks; i++) {
  #pragma unroll
        for (int j = 0; j < 4; j++) {
          int wr = c_sh_wr + 8 * j;
          write(wr + (4 * c_sh_stride) * 0 + 0, frag_c[i][j][0][0],
                frag_c[i][j][0][1], frag_s[j / 2][2 * (j % 2) + 0]);
          write(wr + (4 * c_sh_stride) * 8 + 0, frag_c[i][j][0][2],
                frag_c[i][j][0][3], frag_s[j / 2][2 * (j % 2) + 0]);
          write(wr + (4 * c_sh_stride) * 0 + 4, frag_c[i][j][1][0],
                frag_c[i][j][1][1], frag_s[j / 2][2 * (j % 2) + 1]);
          write(wr + (4 * c_sh_stride) * 8 + 4, frag_c[i][j][1][2],
                frag_c[i][j][1][3], frag_s[j / 2][2 * (j % 2) + 1]);
        }
        c_sh_wr += 16 * (4 * c_sh_stride);
      }
    }
    __syncthreads();

  #pragma unroll
    for (int i = 0;
         i < div_ceil(16 * thread_m_blocks, threads / (2 * thread_n_blocks));
         i++) {
      if (c_gl_wr < c_gl_wr_end) {
        C[c_gl_wr] = sh_red[c_sh_rd];
        c_gl_wr += c_gl_wr_delta;
        c_sh_rd += c_sh_rd_delta;
      }
    }
  };

  // Start global fetch and register load pipelines.
  auto start_pipes = [&]() {

  #pragma unroll
    for (int i = 0; i < stages - 1; i++) {
      if (has_act_order && i == 0) {
        int last_g_idx = slice_k_start + stages * tb_k * 2;
        if (last_g_idx >= prob_k) {
          last_g_idx = prob_k - 1;
        }
        fetch_scales_to_shared(true, g_idx[slice_k_start], g_idx[last_g_idx]);
      }

      if constexpr (has_zp && group_blocks == -1) {
        if (i == 0) {
          fetch_zp_to_shared();
        }
      }
      fetch_to_shared(i, i, i < slice_iters);
    }

    zero_accums();
    wait_for_stage();
    init_same_group(0);
    fetch_to_registers(0, 0);
    fetch_scales_to_registers(0, 0);
    if constexpr (has_zp) {
      fetch_zp_to_registers(0, 0);
    }
    a_gl_rd += a_gl_rd_delta_o * (stages - 1);
    slice_k_start_shared_fetch += tb_k * (stages - 1);
  };
  if (slice_iters) {
    start_pipes();
  }

  // Main loop.
  while (slice_iters) {
    // We unroll over both the global fetch and the register load pipeline to
    // ensure all shared memory accesses are static. Note that both pipelines
    // have even length meaning that the next iteration will always start at
    // index 0.

  #pragma unroll
    for (int pipe = 0; pipe < stages;) {
  #pragma unroll
      for (int k = 0; k < b_sh_wr_iters; k++) {
        fetch_to_registers(k + 1, pipe % stages);
        fetch_scales_to_registers(k + 1, pipe);
        if constexpr (has_zp) {
          fetch_zp_to_registers(k + 1, pipe);
        }
        if (k == b_sh_wr_iters - 2) {
          fetch_to_shared((pipe + stages - 1) % stages, pipe,
                          slice_iters >= stages);
          pipe++;
          wait_for_stage();
          init_same_group(pipe % stages);
        }
        matmul(k);
      }
      slice_iters--;
      if (slice_iters == 0) {
        break;
      }
    }

    a_gl_rd += a_gl_rd_delta_o * stages;
    slice_k_start += tb_k * stages;
    slice_k_start_shared_fetch += tb_k * stages;

    if constexpr (has_act_order) {
      int first_group_id = g_idx[slice_k_start];
      int last_g_idx = slice_k_start + stages * tb_k * 2;
      if (last_g_idx >= prob_k) {
        last_g_idx = prob_k - 1;
      }
      int last_group_id = g_idx[last_g_idx];
      if (last_group_id >= sh_first_group_id + sh_num_groups) {
        fetch_scales_to_shared(false, first_group_id, last_group_id);
        __syncthreads();
      }
    }

    // Process results and, if necessary, proceed to the next column slice.
    // While this pattern may not be the most readable, other ways of writing
    // the loop seemed to noticeably worse performance after compilation.
    if (slice_iters == 0) {
      cp_async_wait<0>();
      bool last = slice_idx == slice_count - 1;
      // For per-column scales, we only fetch them here in the final step before
      // write-out
      if (group_blocks == -1 && last) {
        if (s_sh_wr_pred) cp_async4(&sh_s[s_sh_wr], &scales_ptr[s_gl_rd]);
        cp_async_fence();
      }
      thread_block_reduce();
      if (group_blocks == -1 && last) {
        cp_async_wait<0>();
        __syncthreads();
        if (threadIdx.x / 32 < thread_n_blocks / 4) {
          reinterpret_cast<int4*>(&frag_s)[0] = sh_s[s_sh_rd + 0];
          reinterpret_cast<int4*>(&frag_s)[1] = sh_s[s_sh_rd + 4];
        }
      }
      if (slice_count > 1) {  // only globally reduce if there is more than one
                              // block in a slice
        barrier_acquire(&locks[slice_col], slice_idx);
        global_reduce(slice_idx == 0, last);
        barrier_release(&locks[slice_col], last);
      }
      if (last)  // only the last block in a slice actually writes the result
        write_result();
      slice_row = 0;
      slice_col_par++;
      slice_col++;
      init_slice();
      if (slice_iters) {
        a_gl_rd = a_gl_stride * (threadIdx.x / a_gl_rd_delta_o) +
                  (threadIdx.x % a_gl_rd_delta_o);
  #pragma unroll
        for (int i = 0; i < b_sh_wr_iters; i++)
          B_ptr[i] += b_sh_stride - b_gl_rd_delta_o * k_tiles;
        if (slice_col == 0) {
  #pragma unroll
          for (int i = 0; i < b_sh_wr_iters; i++) B_ptr[i] -= b_gl_stride;
        }

        // Update slice k/n for scales loading
        if constexpr (has_act_order) {
          slice_k_start = tb_k * slice_row;
          slice_k_finish = slice_k_start + tb_k * slice_iters;
          slice_k_start_shared_fetch = slice_k_start;
          slice_n_offset = act_s_col_tb_stride * slice_col;

        } else {
          s_gl_rd = s_sh_stride * slice_col + threadIdx.x;
          zp_gl_rd = zp_sh_stride * slice_col + threadIdx.x;
        }

        start_pipes();
      }
    }
  }
}


// 8 warps are a good choice since every SM has 4 schedulers and having more
// than 1 warp per schedule allows some more latency hiding. At the same time,
// we want relatively few warps to have many registers per warp and small tiles.
const int USER_THREADS = 256; // Note: This is only used with user-provided thread_k/n
const int STAGES = 4;  // 4 pipeline stages fit into shared memory
const int SHARED_MEM = 96 * 1024;  // max shared memory on compute capability 8.6 (< 8.0)
static constexpr int pack_factor_4bit = 8;  // We have 8 4-bit vals inside a 32 bit

#define __CALL_IF(THREAD_M_BLOCKS, THREAD_N_BLOCKS, THREAD_K_BLOCKS,           \
                  GROUP_BLOCKS, NUM_THREADS)                                   \
  else if (thread_m_blocks == THREAD_M_BLOCKS &&                               \
           thread_n_blocks == THREAD_N_BLOCKS &&                               \
           thread_k_blocks == THREAD_K_BLOCKS &&                               \
           group_blocks == GROUP_BLOCKS && num_threads == NUM_THREADS) {       \
    hipFuncSetAttribute(reinterpret_cast<const void*>(Marlin<scalar_t), NUM_THREADS, THREAD_M_BLOCKS, THREAD_N_BLOCKS, \
                                THREAD_K_BLOCKS, STAGES, GROUP_BLOCKS, w_type_id, false, has_zp, num_bits>,        \
                         hipFuncAttributeMaxDynamicSharedMemorySize,          \
                         SHARED_MEM);                                          \
    Marlin<scalar_t, NUM_THREADS, THREAD_M_BLOCKS, THREAD_N_BLOCKS, THREAD_K_BLOCKS,     \
           STAGES, GROUP_BLOCKS, w_type_id, false, has_zp, num_bits><<<blocks, NUM_THREADS, SHARED_MEM, stream>>>( \
        A_ptr, B_ptr, C_ptr, s_ptr, zp_ptr, g_idx_ptr, prob_m, prob_n, prob_k, num_groups, locks);            \
  }

typedef struct {
  int thread_k;
  int thread_n;
  int num_threads;
} thread_config_t;

typedef struct {
  int max_m_blocks;
  thread_config_t tb_cfg;
} exec_config_t;

thread_config_t small_batch_thread_configs[] = {
    // Ordered by priority
    // thread_k, thread_n, num_threads
    {128, 128, 256}, // Default
    {128, 64, 128},  // Reduce N 2X, same K
    {64, 256, 256},  // Reduce K 2X, increase N 2X
    {64, 128, 128},  // Reduce K 2X, same N
};

thread_config_t large_batch_thread_configs[] = {
    // Ordered by priority
    // thread_k, thread_n, num_threads
    {64, 256, 256},  // Default
    {128, 128, 256}, // Reduce N 2X, increase K 2X
    {64, 128, 128},  // Reduce N 2X, same K
    {128, 64, 128},  // Reduce N 4X, increase K 2X
};

int get_scales_cache_size(thread_config_t const& th_config, int prob_m,
                          int prob_n, int prob_k, int num_bits, int group_size,
                          bool has_act_order, bool is_k_full) {
  bool cache_scales_chunk = has_act_order && !is_k_full;

  int tb_n = th_config.thread_n;
  int tb_k = th_config.thread_k;

  // Get max scale groups per thread-block
  int tb_groups;
  if (group_size == -1) {
    tb_groups = 1;
  } else if (group_size == 0) {
    tb_groups = div_ceil(tb_k, 32);  // Worst case is 32 group size
  } else {
    tb_groups = div_ceil(tb_k, group_size);
  }

  if (cache_scales_chunk) {
    int load_groups =
        tb_groups * pipe_stages * 2;     // Chunk size is 2x pipeline over dim K
    load_groups = max(load_groups, 32);  // We load at least 32 scale groups
    return load_groups * tb_n * 2;

  } else {
    int tb_scales = tb_groups * tb_n * 2;

    return tb_scales * pipe_stages;
  }
}

bool is_valid_cache_size(thread_config_t const& th_config, int max_m_blocks,
                         int prob_m, int prob_n, int prob_k, int num_bits,
                         int scales_cache_size, int max_shared_mem) {
  int pack_factor = 32 / num_bits;

  // Get B size
  int tb_k = th_config.thread_k;
  int tb_n = th_config.thread_n;

  int b_size = (tb_k * tb_n / pack_factor) * 4;

  // Get A size
  int m_blocks = div_ceil(prob_m, 16);
  int tb_max_m = 16;

  while (true) {
    if (m_blocks >= max_m_blocks) {
      tb_max_m *= max_m_blocks;
      break;
    }

    max_m_blocks--;
    if (max_m_blocks == 0) {
      CHECK(false, "Unexpected m_blocks = ", m_blocks);
    }
  }

  int a_size = (tb_max_m * tb_k) * 2;

  float pipe_size = (a_size + b_size) * pipe_stages;

  float reduce_size = max(th_config.num_threads * 32 * 4,
                          (tb_n / 64) * 32 * (tb_max_m / 16) * 4 * 2 * 4 * 2);

  CHECK(max_shared_mem / 2 > scales_cache_size);  // Sanity

  return pipe_size + reduce_size < 0.95f * (max_shared_mem - scales_cache_size);
}

bool is_valid_config(thread_config_t const& th_config, int max_m_blocks,
                     int prob_m, int prob_n, int prob_k, int num_bits,
                     int group_size, bool has_act_order, bool is_k_full,
                     int max_shared_mem) {
  // Sanity
  if (th_config.thread_k == -1 || th_config.thread_n == -1 ||
      th_config.num_threads == -1) {
    return false;
  }

  // Verify K/N are divisible by thread K/N
  if (prob_k % th_config.thread_k != 0 || prob_n % th_config.thread_n != 0) {
    return false;
  }

  // Verify min for thread K/N
  if (th_config.thread_n < min_thread_n || th_config.thread_k < min_thread_k) {
    return false;
  }

  // num_threads must be at least 128 (= 4 warps)
  if (th_config.num_threads < 128) {
    return false;
  }

  //  Determine cache for scales
  int scales_cache_size =
      get_scales_cache_size(th_config, prob_m, prob_n, prob_k, num_bits,
                            group_size, has_act_order, is_k_full);

  // Check that pipeline fits into cache
  if (!is_valid_cache_size(th_config, max_m_blocks, prob_m, prob_n, prob_k,
                           num_bits, scales_cache_size, max_shared_mem)) {
    return false;
  }

  return true;
}

exec_config_t determine_thread_config(int prob_m, int prob_n, int prob_k,
                                      int num_bits, int group_size,
                                      bool has_act_order, bool is_k_full,
                                      int max_shared_mem) {
  int max_m_blocks = 4;
  while (max_m_blocks > 0) {
    if (prob_m <= 16) {
      for (auto th_config : small_batch_thread_configs) {
        if (is_valid_config(th_config, max_m_blocks, prob_m, prob_n, prob_k,
                            num_bits, group_size, has_act_order, is_k_full,
                            max_shared_mem)) {
          return exec_config_t{max_m_blocks, th_config};
        }
      }
    } else {
      for (auto th_config : large_batch_thread_configs) {
        if (is_valid_config(th_config, max_m_blocks, prob_m, prob_n, prob_k,
                            num_bits, group_size, has_act_order, is_k_full,
                            max_shared_mem)) {
          return exec_config_t{max_m_blocks, th_config};
        }
      }
    }

    max_m_blocks--;  // Process less M blocks per invocation to reduce cache
                     // usage
  }

  return exec_config_t{0, {-1, -1, -1}};
}

#define CALL_IF(N_BLOCKS, K_BLOCKS, NUM_THREADS)    \
  __CALL_IF(1, N_BLOCKS, K_BLOCKS, -1, NUM_THREADS) \
  __CALL_IF(1, N_BLOCKS, K_BLOCKS, 4, NUM_THREADS)  \
  __CALL_IF(1, N_BLOCKS, K_BLOCKS, 8, NUM_THREADS)  \
  __CALL_IF(2, N_BLOCKS, K_BLOCKS, -1, NUM_THREADS) \
  __CALL_IF(2, N_BLOCKS, K_BLOCKS, 4, NUM_THREADS)  \
  __CALL_IF(2, N_BLOCKS, K_BLOCKS, 8, NUM_THREADS)  \
  __CALL_IF(3, N_BLOCKS, K_BLOCKS, -1, NUM_THREADS) \
  __CALL_IF(3, N_BLOCKS, K_BLOCKS, 4, NUM_THREADS)  \
  __CALL_IF(3, N_BLOCKS, K_BLOCKS, 8, NUM_THREADS)  \
  __CALL_IF(4, N_BLOCKS, K_BLOCKS, -1, NUM_THREADS) \
  __CALL_IF(4, N_BLOCKS, K_BLOCKS, 4, NUM_THREADS) \
  __CALL_IF(4, N_BLOCKS, K_BLOCKS, 8, NUM_THREADS)

template<typename scalar_t,
          const ScalarTypeID w_type_id,
          const bool has_zp,            // whether zero-points are enabled
          const int num_bits
>
void marlin_matmul(const void* A, const void* B, void* scales, void* zeros, void* C, int prob_m, int prob_k, 
                 int prob_n, void* workspace, int groupsize, int64_t stream_
                 ) {
  
  int dev = 0; 
  hipStream_t stream = (hipStream_t)stream_; 
  int thread_k = -1;
  int thread_n = -1; 
  int sms = -1; 
  int max_par = 16;

  int tot_m = prob_m;
  int tot_m_blocks = div_ceil(tot_m, 16);
  int pad = 16 * tot_m_blocks - tot_m;

  bool has_act_order = false;
  bool is_k_full = true;
  if (sms == -1)
    hipDeviceGetAttribute(&sms, hipDeviceAttributeMultiprocessorCount, dev);
  int max_shared_mem = 0;
  hipDeviceGetAttribute(&max_shared_mem,
                         hipDeviceAttributeSharedMemPerBlockOptin, 0);
  CHECK(max_shared_mem > 0, "error");
  // Set thread config
  exec_config_t exec_cfg;
  if (thread_k != -1 && thread_n != -1) {
    // User-defined config
    exec_cfg =
        exec_config_t{4, thread_config_t{thread_k, thread_n, default_threads}};
  } else {
    // Auto config
    exec_cfg =
        determine_thread_config(prob_m, prob_n, prob_k, num_bits, groupsize,
                                has_act_order, is_k_full, max_shared_mem);
  }

  int num_threads = exec_cfg.tb_cfg.num_threads;
  thread_k = exec_cfg.tb_cfg.thread_k;
  thread_n = exec_cfg.tb_cfg.thread_n;

  int thread_k_blocks = thread_k / 16;
  int thread_n_blocks = thread_n / 16;
  int group_blocks = (groupsize == -1) ? -1 : groupsize / 16;
  int blocks = sms;
  int num_groups = prob_k / groupsize;

  if (prob_m == 0 || prob_n == 0 || prob_k == 0) {
    return;
  }

  const int4* A_ptr = (const int4*)A;
  const int4* B_ptr = (const int4*)B;
  int4* C_ptr = (int4*)C;
  const int4* s_ptr = (const int4*)scales;
  const int4* zp_ptr = (const int4*)zeros;
  const int* g_idx_ptr = (const int*)nullptr;
  int* locks = (int*)workspace;


  // if (has_act_order) {
  //   // Permute A columns
  //   int block_rows = div_ceil(prob_m, blocks);
  //   permute_cols_kernel<<<blocks, default_threads, 0, stream>>>(
  //       A_ptr, perm_ptr, a_tmp_ptr, prob_m, prob_k, prob_k, block_rows);
  //   A_ptr = a_tmp_ptr;
  // }

  // // If we have a full K, then we can run the non-act-order version of Marlin
  // // (since the weight rows are reordered by increasing group ids, and by having
  // // a full K, we have full original groups)
  // if (is_k_full) {
  //   has_act_order = false;
  // }

  for (int i = 0; i < tot_m_blocks; i += exec_cfg.max_m_blocks) {
    int thread_m_blocks = tot_m_blocks - i;
    prob_m = tot_m - 16 * i;
    int par = 1;
    if (thread_m_blocks > exec_cfg.max_m_blocks) {
      // Note that parallel > 1 currently only works for inputs without any
      // padding
      par = (16 * thread_m_blocks - pad) / (16 * exec_cfg.max_m_blocks);
      if (par > max_par) par = max_par;
      prob_m = (16 * exec_cfg.max_m_blocks) * par;
      i += exec_cfg.max_m_blocks * (par - 1);
      thread_m_blocks = exec_cfg.max_m_blocks;
    }


    // For compilation speed, we only define the kernel configurations that have
    // seemed useful (in terms of performance) in our testing, however many more
    // are, in principle, possible.
    if (false) {
    }
    CALL_IF(8, 8, 256)
    CALL_IF(16, 4, 256)
    CALL_IF(8, 4, 128)
    CALL_IF(4, 8, 128)
    else {
      throw std::runtime_error("Unsupported shapes: MKN");
    }

    A_ptr += 16 * thread_m_blocks * (prob_k / 8) * par;
    C_ptr += 16 * thread_m_blocks * (prob_n / 8) * par;
  }
}

extern "C" void marlin_gptq_4bit_f16(const void* A, const void* B, void* scales, void* zeros, void* C, int prob_m, int prob_k, 
                 int prob_n, void* workspace, int groupsize, int64_t stream
                 ) {
    marlin_matmul<half, ScalarTypeID::kU4B8, false, 4>(A, B, scales, zeros, C, prob_m, prob_k, prob_n, workspace, groupsize, stream);
}

extern "C" void marlin_gptq_4bit_bf16(const void* A, const void* B, void* scales, void* zeros, void* C, int prob_m, int prob_k, 
                 int prob_n, void* workspace, int groupsize, int64_t stream
                 ) {
    marlin_matmul<hip_bfloat16, ScalarTypeID::kU4B8, false, 4>(A, B, scales, zeros, C, prob_m, prob_k, prob_n, workspace, groupsize, stream);
}

extern "C" void marlin_awq_4bit_f16(const void* A, const void* B, void* scales, void* zeros, void* C, int prob_m, int prob_k, 
                 int prob_n, void* workspace, int groupsize, int64_t stream
                 ) {
    marlin_matmul<half, ScalarTypeID::kU4, true, 4>(A, B, scales, zeros, C, prob_m, prob_k, prob_n, workspace, groupsize, stream);
}

extern "C" void marlin_awq_4bit_bf16(const void* A, const void* B, void* scales, void* zeros, void* C, int prob_m, int prob_k, 
                 int prob_n, void* workspace, int groupsize, int64_t stream
                 ) {
    marlin_matmul<hip_bfloat16, ScalarTypeID::kU4, true, 4>(A, B, scales, zeros, C, prob_m, prob_k, prob_n, workspace, groupsize, stream);
}


template <int const num_threads, int const num_bits, bool const has_perm>
__global__ void
gptq_marlin_repack_kernel(uint32_t const *__restrict__ b_q_weight_ptr,
                          uint32_t const *__restrict__ perm_ptr,
                          uint32_t *__restrict__ out_ptr, int size_k,
                          int size_n) {
  constexpr int pack_factor = 32 / num_bits;

  int k_tiles = size_k / tile_k_size;
  int n_tiles = size_n / tile_n_size;
  int block_k_tiles = div_ceil(k_tiles, gridDim.x);

  int start_k_tile = blockIdx.x * block_k_tiles;
  if (start_k_tile >= k_tiles) {
    return;
  }

  int finish_k_tile = min(start_k_tile + block_k_tiles, k_tiles);

  // Wait until the next thread tile has been loaded to shared memory.
  auto wait_for_stage = [&]() {
    // We only have `stages - 2` active fetches since we are double buffering
    // and can only issue the next fetch when it is guaranteed that the previous
    // shared memory load is fully complete (as it may otherwise be
    // overwritten).
    cp_async_wait<repack_stages - 2>();
    __syncthreads();
  };

  extern __shared__ int4 sh[];

  constexpr int perm_size = tile_k_size / 4;

  int4 *sh_perm_ptr = sh;
  int4 *sh_pipe_ptr = sh_perm_ptr;
  if constexpr (has_perm) {
    sh_pipe_ptr += perm_size;
  }

  constexpr int tile_ints = tile_k_size / pack_factor;

  constexpr int stage_n_threads = tile_n_size / 4;
  constexpr int stage_k_threads = has_perm ? tile_k_size : tile_ints;
  constexpr int stage_size = stage_k_threads * stage_n_threads;

  auto load_perm_to_shared = [&](int k_tile_id) {
    int first_k_int4 = (k_tile_id * tile_k_size) / 4;

    int4 const *perm_int4_ptr = reinterpret_cast<int4 const *>(perm_ptr);

    if (threadIdx.x < perm_size) {
      sh_perm_ptr[threadIdx.x] = perm_int4_ptr[first_k_int4 + threadIdx.x];
    }
    __syncthreads();
  };

  auto fetch_to_shared = [&](int pipe, int k_tile_id, int n_tile_id) {
    if (n_tile_id >= n_tiles) {
      cp_async_fence();
      return;
    }

    int first_n = n_tile_id * tile_n_size;

    int4 *sh_ptr = sh_pipe_ptr + stage_size * pipe;

    if constexpr (has_perm) {
      if (threadIdx.x < stage_size) {
        int k_id = threadIdx.x / stage_n_threads;
        int n_id = threadIdx.x % stage_n_threads;

        uint32_t const *sh_perm_int_ptr =
            reinterpret_cast<uint32_t const *>(sh_perm_ptr);

        int src_k = sh_perm_int_ptr[k_id];
        int src_k_packed = src_k / pack_factor;

        cp_async4(
            &sh_ptr[k_id * stage_n_threads + n_id],
            reinterpret_cast<int4 const *>(&(
                b_q_weight_ptr[src_k_packed * size_n + first_n + (n_id * 4)])));
      }

    } else {
      if (threadIdx.x < stage_size) {
        int k_id = threadIdx.x / stage_n_threads;
        int n_id = threadIdx.x % stage_n_threads;

        int first_k = k_tile_id * tile_k_size;
        int first_k_packed = first_k / pack_factor;

        cp_async4(&sh_ptr[k_id * stage_n_threads + n_id],
                  reinterpret_cast<int4 const *>(
                      &(b_q_weight_ptr[(first_k_packed + k_id) * size_n +
                                       first_n + (n_id * 4)])));
      }
    }

    cp_async_fence();
  };

  auto repack_tile = [&](int pipe, int k_tile_id, int n_tile_id) {
    if (n_tile_id >= n_tiles) {
      return;
    }

    int warp_id = threadIdx.x / 32;
    int th_id = threadIdx.x % 32;

    if (warp_id >= 4) {
      return;
    }

    int tc_col = th_id / 4;
    int tc_row = (th_id % 4) * 2;

    constexpr int tc_offsets[4] = {0, 1, 8, 9};

    int cur_n = warp_id * 16 + tc_col;

    constexpr int sh_stride = 64;
    constexpr uint32_t mask = (1 << num_bits) - 1;

    int4 *sh_stage_ptr = sh_pipe_ptr + stage_size * pipe;
    uint32_t *sh_stage_int_ptr = reinterpret_cast<uint32_t *>(sh_stage_ptr);

    uint32_t *sh_perm_int_ptr = reinterpret_cast<uint32_t *>(sh_perm_ptr);

    uint32_t vals[8];

    if constexpr (has_perm) {
      for (int i = 0; i < 4; i++) {
        int k_idx = tc_row + tc_offsets[i];

        uint32_t src_k = sh_perm_int_ptr[k_idx];
        uint32_t src_k_pos = src_k % pack_factor;

        uint32_t b1_val = sh_stage_int_ptr[k_idx * sh_stride + cur_n];
        uint32_t b1_cur_val = (b1_val >> (src_k_pos * num_bits)) & mask;

        uint32_t b2_val = sh_stage_int_ptr[k_idx * sh_stride + cur_n + 8];
        uint32_t b2_cur_val = (b2_val >> (src_k_pos * num_bits)) & mask;

        vals[i] = b1_cur_val;
        vals[4 + i] = b2_cur_val;
      }

    } else {
      uint32_t b1_vals[tile_ints];
      uint32_t b2_vals[tile_ints];

#pragma unroll
      for (int i = 0; i < tile_ints; i++) {
        b1_vals[i] = sh_stage_int_ptr[cur_n + sh_stride * i];
        b2_vals[i] = sh_stage_int_ptr[cur_n + 8 + sh_stride * i];
      }

#pragma unroll
      for (int i = 0; i < 4; i++) {
        int cur_elem = tc_row + tc_offsets[i];
        int cur_int = cur_elem / pack_factor;
        int cur_pos = cur_elem % pack_factor;

        vals[i] = (b1_vals[cur_int] >> (cur_pos * num_bits)) & mask;
        vals[4 + i] = (b2_vals[cur_int] >> (cur_pos * num_bits)) & mask;
      }
    }

    constexpr int tile_size = tile_k_size * tile_n_size / pack_factor;
    int out_offset = (k_tile_id * n_tiles + n_tile_id) * tile_size;

    // Result of:
    // https://github.com/NVIDIA/FasterTransformer/blob/main/src/fastertransformer/cutlass_extensions/include/cutlass_extensions/interleaved_numeric_conversion.h
    if constexpr (num_bits == 4) {
      constexpr int pack_idx[8] = {0, 2, 4, 6, 1, 3, 5, 7};

      uint32_t res = 0;
#pragma unroll
      for (int i = 0; i < 8; i++) {
        res |= vals[pack_idx[i]] << (i * 4);
      }

      out_ptr[out_offset + th_id * 4 + warp_id] = res;

    } else {
      constexpr int pack_idx[4] = {0, 2, 1, 3};

      uint32_t res1 = 0;
      uint32_t res2 = 0;
#pragma unroll
      for (int i = 0; i < 4; i++) {
        res1 |= vals[pack_idx[i]] << (i * 8);
        res2 |= vals[4 + pack_idx[i]] << (i * 8);
      }

      out_ptr[out_offset + th_id * 8 + (warp_id * 2) + 0] = res1;
      out_ptr[out_offset + th_id * 8 + (warp_id * 2) + 1] = res2;
    }
  };

  auto start_pipes = [&](int k_tile_id, int n_tile_id) {
#pragma unroll
    for (int pipe = 0; pipe < repack_stages - 1; pipe++) {
      fetch_to_shared(pipe, k_tile_id, n_tile_id + pipe);
    }

    wait_for_stage();
  };
#pragma unroll
  for (int k_tile_id = start_k_tile; k_tile_id < finish_k_tile; k_tile_id++) {
    int n_tile_id = 0;

    if constexpr (has_perm) {
      load_perm_to_shared(k_tile_id);
    }

    start_pipes(k_tile_id, n_tile_id);

    while (n_tile_id < n_tiles) {
#pragma unroll
      for (int pipe = 0; pipe < repack_stages; pipe++) {
        fetch_to_shared((pipe + repack_stages - 1) % repack_stages, k_tile_id,
                        n_tile_id + pipe + repack_stages - 1);
        repack_tile(pipe, k_tile_id, n_tile_id + pipe);
        wait_for_stage();
      }
      n_tile_id += repack_stages;
    }
  }
}

#define CALL_IF2(NUM_BITS, HAS_PERM)                                           \
  else if (num_bits == NUM_BITS && has_perm == HAS_PERM) {                     \
    hipFuncSetAttribute(reinterpret_cast<const void*>(                                                      \
        gptq_marlin_repack_kernel<repack_threads), NUM_BITS, HAS_PERM>,         \
        hipFuncAttributeMaxDynamicSharedMemorySize, max_shared_mem);          \
    gptq_marlin_repack_kernel<repack_threads, NUM_BITS, HAS_PERM>              \
        <<<blocks, repack_threads, max_shared_mem, stream>>>(                  \
            b_q_weight_ptr, perm_ptr, out_ptr, size_k, size_n);                \
  }

extern "C" void gptq_marlin_repack(void *weight, void *perm, void *out,
                                   int size_k, int size_n, int num_bits, int64_t stream_) {
  // Verify compatibility with marlin tile of 16x64
  assert(size_k % tile_k_size == 0);
  assert(size_n % tile_n_size == 0);
  assert(num_bits == 4 || num_bits == 8);
  const int pack_factor = 32 / num_bits;

  bool has_perm = true;
  int dev = 0;
  // Get ptrs
  uint32_t const *b_q_weight_ptr = reinterpret_cast<uint32_t const *>(weight);
  uint32_t const *perm_ptr = reinterpret_cast<uint32_t const *>(perm);
  uint32_t *out_ptr = reinterpret_cast<uint32_t *>(out);

  // Get dev info
  hipStream_t stream = (hipStream_t)stream_;
  int blocks;
  hipDeviceGetAttribute(&blocks, hipDeviceAttributeMultiprocessorCount, dev);

  int max_shared_mem = 0;
  hipDeviceGetAttribute(&max_shared_mem,
                         hipDeviceAttributeSharedMemPerBlockOptin, dev);
  assert(max_shared_mem > 0);

  if (false) {
  }
  CALL_IF2(4, false)
  CALL_IF2(4, true)
  CALL_IF2(8, false)
  CALL_IF2(8, true)
  else {
    assert(false);
  }
}


template <int const num_threads, int const num_bits>
__global__ void awq_marlin_repack_kernel(
    uint32_t const* __restrict__ b_q_weight_ptr, uint32_t* __restrict__ out_ptr,
    int size_k, int size_n) {
  constexpr int pack_factor = 32 / num_bits;

  int k_tiles = size_k / tile_k_size;
  int n_tiles = size_n / tile_n_size;
  int block_k_tiles = div_ceil(k_tiles, gridDim.x);

  auto start_k_tile = blockIdx.x * block_k_tiles;
  if (start_k_tile >= k_tiles) {
    return;
  }

  int finish_k_tile = min(start_k_tile + block_k_tiles, k_tiles);

  // Wait until the next thread tile has been loaded to shared memory.
  auto wait_for_stage = [&]() {
    // We only have `stages - 2` active fetches since we are double buffering
    // and can only issue the next fetch when it is guaranteed that the previous
    // shared memory load is fully complete (as it may otherwise be
    // overwritten).
    cp_async_wait<repack_stages - 2>();
    __syncthreads();
  };

  extern __shared__ int4 sh[];

  constexpr int tile_n_ints = tile_n_size / pack_factor;

  constexpr int stage_n_threads = tile_n_ints / 4;
  constexpr int stage_k_threads = tile_k_size;
  constexpr int stage_size = stage_k_threads * stage_n_threads;

  auto fetch_to_shared = [&](int pipe, int k_tile_id, int n_tile_id) {
    if (n_tile_id >= n_tiles) {
      cp_async_fence();
      return;
    }

    int first_n = n_tile_id * tile_n_size;
    int first_n_packed = first_n / pack_factor;

    int4* sh_ptr = sh + stage_size * pipe;

    if (threadIdx.x < stage_size) {
      auto k_id = threadIdx.x / stage_n_threads;
      auto n_id = threadIdx.x % stage_n_threads;

      int first_k = k_tile_id * tile_k_size;

      cp_async4(&sh_ptr[k_id * stage_n_threads + n_id],
                reinterpret_cast<int4 const*>(
                    &(b_q_weight_ptr[(first_k + k_id) * (size_n / pack_factor) +
                                     first_n_packed + (n_id * 4)])));
    }

    cp_async_fence();
  };

  auto repack_tile = [&](int pipe, int k_tile_id, int n_tile_id) {
    if (n_tile_id >= n_tiles) {
      return;
    }

    auto warp_id = threadIdx.x / 32;
    auto th_id = threadIdx.x % 32;

    if (warp_id >= 4) {
      return;
    }

    int tc_col = th_id / 4;
    int tc_row = (th_id % 4) * 2;

    constexpr int tc_offsets[4] = {0, 1, 8, 9};

    int cur_n = warp_id * 16 + tc_col;
    int cur_n_packed = cur_n / pack_factor;
    int cur_n_pos = cur_n % pack_factor;

    constexpr int sh_stride = tile_n_ints;
    constexpr uint32_t mask = (1 << num_bits) - 1;

    int4* sh_stage_ptr = sh + stage_size * pipe;
    uint32_t* sh_stage_int_ptr = reinterpret_cast<uint32_t*>(sh_stage_ptr);

    // Undo interleaving
    int cur_n_pos_unpacked;
    if constexpr (num_bits == 4) {
      constexpr int undo_pack[8] = {0, 4, 1, 5, 2, 6, 3, 7};
      cur_n_pos_unpacked = undo_pack[cur_n_pos];
    } else {
      constexpr int undo_pack[4] = {0, 2, 1, 3};
      cur_n_pos_unpacked = undo_pack[cur_n_pos];
    }

    uint32_t vals[8];
#pragma unroll
    for (int i = 0; i < 4; i++) {
      int cur_elem = tc_row + tc_offsets[i];

      int packed_src_0 = sh_stage_int_ptr[cur_n_packed + sh_stride * cur_elem];
      int packed_src_1 = sh_stage_int_ptr[cur_n_packed + (8 / pack_factor) +
                                          sh_stride * cur_elem];

      vals[i] = (packed_src_0 >> (cur_n_pos_unpacked * num_bits)) & mask;
      vals[4 + i] = (packed_src_1 >> (cur_n_pos_unpacked * num_bits)) & mask;
    }

    constexpr int tile_size = tile_k_size * tile_n_size / pack_factor;
    int out_offset = (k_tile_id * n_tiles + n_tile_id) * tile_size;

    // Result of:
    // https://github.com/NVIDIA/FasterTransformer/blob/main/src/fastertransformer/cutlass_extensions/include/cutlass_extensions/interleaved_numeric_conversion.h
    if constexpr (num_bits == 4) {
      constexpr int pack_idx[8] = {0, 2, 4, 6, 1, 3, 5, 7};

      uint32_t res = 0;
#pragma unroll
      for (int i = 0; i < 8; i++) {
        res |= vals[pack_idx[i]] << (i * 4);
      }

      out_ptr[out_offset + th_id * 4 + warp_id] = res;

    } else {
      constexpr int pack_idx[4] = {0, 2, 1, 3};

      uint32_t res1 = 0;
      uint32_t res2 = 0;
#pragma unroll
      for (int i = 0; i < 4; i++) {
        res1 |= vals[pack_idx[i]] << (i * 8);
        res2 |= vals[4 + pack_idx[i]] << (i * 8);
      }

      out_ptr[out_offset + th_id * 8 + (warp_id * 2) + 0] = res1;
      out_ptr[out_offset + th_id * 8 + (warp_id * 2) + 1] = res2;
    }
  };

  auto start_pipes = [&](int k_tile_id, int n_tile_id) {
#pragma unroll
    for (int pipe = 0; pipe < repack_stages - 1; pipe++) {
      fetch_to_shared(pipe, k_tile_id, n_tile_id + pipe);
    }

    wait_for_stage();
  };
#pragma unroll
  for (int k_tile_id = start_k_tile; k_tile_id < finish_k_tile; k_tile_id++) {
    int n_tile_id = 0;

    start_pipes(k_tile_id, n_tile_id);

    while (n_tile_id < n_tiles) {
#pragma unroll
      for (int pipe = 0; pipe < repack_stages; pipe++) {
        fetch_to_shared((pipe + repack_stages - 1) % repack_stages, k_tile_id,
                        n_tile_id + pipe + repack_stages - 1);
        repack_tile(pipe, k_tile_id, n_tile_id + pipe);
        wait_for_stage();
      }
      n_tile_id += repack_stages;
    }
  }
}

#define CALL_IF3(NUM_BITS)                                                   \
  else if (num_bits == NUM_BITS) {                                          \
    hipFuncSetAttribute(reinterpret_cast<const void*>(                                                   \
        awq_marlin_repack_kernel<repack_threads), NUM_BITS>, \
        hipFuncAttributeMaxDynamicSharedMemorySize, max_shared_mem);       \
    awq_marlin_repack_kernel<repack_threads, NUM_BITS>      \
        <<<blocks, repack_threads, max_shared_mem, stream>>>(       \
            weight_ptr, out_ptr, size_k, size_n);                       \
  }

extern "C" void awq_marlin_repack(void* in, void* perm, void* out, int k,
                                int n, int num_bits, int64_t stream_) {

  //in_dim 4096, out_dim 1024 (/pack_factor)
  //ws shape [4096, 128]
  //out_shape [256, 2048]

   //recover original size_k and size_n
   int const pack_factor = 32 / num_bits;
   int size_k = k;
   int size_n = n * pack_factor;

  // Verify compatibility with marlin tile of 16x64
  CHECK(size_k % marlin::tile_k_size == 0, "size_k = ", size_k,
              " is not divisible by tile_k_size = ", marlin::tile_k_size);
  CHECK(size_n % marlin::tile_n_size == 0, "size_n = ", size_n,
              " is not divisible by tile_n_size = ", marlin::tile_n_size);

  CHECK(num_bits == 4 || num_bits == 8,
              "num_bits must be 4 or 8. Got = ", num_bits);
  hipStream_t stream = (hipStream_t)stream_;

  uint32_t const* weight_ptr =
      reinterpret_cast<uint32_t const*>(in);
  uint32_t* out_ptr = reinterpret_cast<uint32_t*>(out);

  int blocks;
  hipDeviceGetAttribute(&blocks, hipDeviceAttributeMultiprocessorCount, 0);
  int max_shared_mem = 0;
  hipDeviceGetAttribute(&max_shared_mem,
                         hipDeviceAttributeSharedMemPerBlockOptin, 0);
  CHECK(max_shared_mem > 0, "error max_shared_mem");

  if (false) {
  }
  CALL_IF3(4)
  CALL_IF3(8)
  else {
    CHECK(false, "Unsupported repack config: num_bits = ", num_bits);
  }
}

#endif
