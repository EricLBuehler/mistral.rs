#include <cstdint>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>

#define CUDA_CHECK(call)                                                       \
  do {                                                                         \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__,         \
              hipGetErrorString(err));                                        \
      exit(err);                                                               \
    }                                                                          \
  } while (0)

template <typename T>
__global__ void fp8_to_dtype_kernel(
    const __hip_fp8_e4m3_fnuz *__restrict__ input,
    T *__restrict__ output,
    size_t num_elements) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_elements) {
    float val = __half2float(__hip_cvt_fp8_to_halfraw(input[idx].__x, __HIP_E4M3_FNUZ));
    output[idx] = static_cast<T>(val);
  }
}

template <typename T>
__global__ void dtype_to_fp8_kernel(
    const T *__restrict__ input,
    __hip_fp8_e4m3_fnuz *__restrict__ output,
    size_t num_elements) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_elements) {
    float val = static_cast<float>(input[idx]);
    // Clamp to FP8 E4M3 range
    if (val > 448.0f) val = 448.0f;
    if (val < -448.0f) val = -448.0f;
    __half h_val = __float2half(val);
    output[idx].__x = __hip_cvt_halfraw_to_fp8(h_val, __HIP_SATFINITE, __HIP_E4M3_FNUZ);
  }
}

extern "C" void launch_fp8_to_f32_kernel(
    const __hip_fp8_e4m3_fnuz *d_input, float *d_output, size_t num_elements,
    hipStream_t stream) {
  const int block_size = 256;
  const int num_blocks = (num_elements + block_size - 1) / block_size;
  
  fp8_to_dtype_kernel<float><<<num_blocks, block_size, 0, stream>>>(
      d_input, d_output, num_elements);
  CUDA_CHECK(hipGetLastError());
}

extern "C" void launch_fp8_to_f16_kernel(
    const __hip_fp8_e4m3_fnuz *d_input, __half *d_output, size_t num_elements,
    hipStream_t stream) {
  const int block_size = 256;
  const int num_blocks = (num_elements + block_size - 1) / block_size;
  
  fp8_to_dtype_kernel<__half><<<num_blocks, block_size, 0, stream>>>(
      d_input, d_output, num_elements);
  CUDA_CHECK(hipGetLastError());
}

extern "C" void launch_fp8_to_bf16_kernel(
    const __hip_fp8_e4m3_fnuz *d_input, __hip_bfloat16 *d_output, size_t num_elements,
    hipStream_t stream) {
  const int block_size = 256;
  const int num_blocks = (num_elements + block_size - 1) / block_size;
  
  fp8_to_dtype_kernel<__hip_bfloat16><<<num_blocks, block_size, 0, stream>>>(
      d_input, d_output, num_elements);
  CUDA_CHECK(hipGetLastError());
}

extern "C" void launch_f32_to_fp8_kernel(
    const float *d_input, __hip_fp8_e4m3_fnuz *d_output, size_t num_elements,
    hipStream_t stream) {
  const int block_size = 256;
  const int num_blocks = (num_elements + block_size - 1) / block_size;
  
  dtype_to_fp8_kernel<float><<<num_blocks, block_size, 0, stream>>>(
      d_input, d_output, num_elements);
  CUDA_CHECK(hipGetLastError());
}

extern "C" void launch_f16_to_fp8_kernel(
    const __half *d_input, __hip_fp8_e4m3_fnuz *d_output, size_t num_elements,
    hipStream_t stream) {
  const int block_size = 256;
  const int num_blocks = (num_elements + block_size - 1) / block_size;
  
  dtype_to_fp8_kernel<__half><<<num_blocks, block_size, 0, stream>>>(
      d_input, d_output, num_elements);
  CUDA_CHECK(hipGetLastError());
}

extern "C" void launch_bf16_to_fp8_kernel(
    const __hip_bfloat16 *d_input, __hip_fp8_e4m3_fnuz *d_output, size_t num_elements,
    hipStream_t stream) {
  const int block_size = 256;
  const int num_blocks = (num_elements + block_size - 1) / block_size;
  
  dtype_to_fp8_kernel<__hip_bfloat16><<<num_blocks, block_size, 0, stream>>>(
      d_input, d_output, num_elements);
  CUDA_CHECK(hipGetLastError());
}