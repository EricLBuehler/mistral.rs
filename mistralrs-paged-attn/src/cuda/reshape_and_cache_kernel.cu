#include "hip/hip_runtime.h"
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <stdint.h>
#include <stdio.h>

#include "cuda_compat.h"

#include <algorithm>
#include <cassert>
#include <map>
#include <vector>

#define CUDA_CHECK(call)                                                       \
  do {                                                                         \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__,         \
              hipGetErrorString(err));                                        \
      exit(err);                                                               \
    }                                                                          \
  } while (0)

namespace vllm {

template <typename scalar_t>
__global__ void reshape_and_cache_kernel(
    const scalar_t *__restrict__ key,   // [num_tokens, num_heads, head_size]
    const scalar_t *__restrict__ value, // [num_tokens, num_heads, head_size]
    scalar_t *__restrict__ key_cache,   // [num_blocks, num_heads, head_size/x,
                                        // block_size, x]
    scalar_t *__restrict__ value_cache, // [num_blocks, num_heads, head_size,
                                        // block_size]
    const int64_t *__restrict__ slot_mapping, // [num_tokens]
    const int key_stride, const int value_stride, const int num_heads,
    const int head_size, const int block_size, const int x) {
  const int64_t token_idx = blockIdx.x;
  const int64_t slot_idx = slot_mapping[token_idx];
  if (slot_idx < 0) {
    // Padding token that should be ignored.
    return;
  }

  const int64_t block_idx = slot_idx / block_size;
  const int64_t block_offset = slot_idx % block_size;

  const int n = num_heads * head_size;
  for (int i = threadIdx.x; i < n; i += blockDim.x) {
    const int64_t src_key_idx = token_idx * key_stride + i;
    const int64_t src_value_idx = token_idx * value_stride + i;

    const int head_idx = i / head_size;
    const int head_offset = i % head_size;
    const int x_idx = head_offset / x;
    const int x_offset = head_offset % x;

    const int64_t tgt_key_idx =
        block_idx * num_heads * (head_size / x) * block_size * x +
        head_idx * (head_size / x) * block_size * x + x_idx * block_size * x +
        block_offset * x + x_offset;
    const int64_t tgt_value_idx =
        block_idx * num_heads * head_size * block_size +
        head_idx * head_size * block_size + head_offset * block_size +
        block_offset;
    key_cache[tgt_key_idx] = key[src_key_idx];
    value_cache[tgt_value_idx] = value[src_value_idx];
  }
}

#define CALL_RESHAPE_AND_CACHE(T)                                              \
  vllm::reshape_and_cache_kernel<T><<<grid, block, 0, stream>>>(               \
      reinterpret_cast<T *>(key), reinterpret_cast<T *>(value),                \
      reinterpret_cast<T *>(key_cache), reinterpret_cast<T *>(value_cache),    \
      slot_mapping, key_stride, value_stride, num_heads, head_size,            \
      block_size, x);

} // namespace vllm

extern "C" void reshape_and_cache(
    void *key,         // [num_tokens, num_heads, head_size]
    void *value,       // [num_tokens, num_heads, head_size]
    void *key_cache,   // [num_blocks, num_heads, head_size/x, block_size, x]
    void *value_cache, // [num_blocks, num_heads, head_size, block_size]
    int64_t *slot_mapping, // [num_tokens]

    int32_t num_tokens, int32_t num_heads, int32_t head_size,
    int32_t block_size, int32_t x, int32_t key_stride, int32_t value_stride,
    hipStream_t stream,

    uint32_t dtype // 0 => f16; 1 => bf16; 2 => f32
) {
  dim3 grid(num_tokens);
  dim3 block(std::min(num_heads * head_size, 512));

  if (dtype == 0) {
    CALL_RESHAPE_AND_CACHE(uint16_t);
  } else if (dtype == 1) {
    CALL_RESHAPE_AND_CACHE(__hip_bfloat16);
  } else if (dtype == 2) {
    CALL_RESHAPE_AND_CACHE(float);
  }
  CUDA_CHECK(hipGetLastError());
}